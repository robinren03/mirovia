#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <omp.h>
#include <hip/hip_runtime.h>

#define THREADS_PER_DIM 16
#define BLOCKS_PER_DIM 16
#define THREADS_PER_BLOCK THREADS_PER_DIM*THREADS_PER_DIM

#include "cudacommon.h"
#include "ResultDatabase.h"
#include "OptionParser.h"
#include "kmeans_cuda_kernel.cu"

//#define BLOCK_DELTA_REDUCE
//#define BLOCK_CENTER_REDUCE

#define CPU_DELTA_REDUCE
#define CPU_CENTER_REDUCE

int setup(ResultDatabase &resultDB, OptionParser &op);

// GLOBAL!!!!!
<<<<<<< HEAD
/* sqrt(256) -- see references for this choice */
unsigned int num_threads_perdim = THREADS_PER_DIM;					
/* temporary */
unsigned int num_blocks_perdim = BLOCKS_PER_DIM;
/* number of threads */
unsigned int num_threads = num_threads_perdim*num_threads_perdim;	
/* number of blocks */
unsigned int num_blocks = num_blocks_perdim*num_blocks_perdim;		

/* _d denotes it resides on the device */
/* newly assignment membership */
int    *membership_new;												
/* inverted data array */
float  *feature_d;													
/* original (not inverted) data array */
float  *feature_flipped_d;											
/* membership on the device */
int    *membership_d;												
/* sum of points in a cluster (per block) */
float  *block_new_centers;											
/* cluster centers on the device */
float  *clusters_d;													
/* per block calculation of cluster centers */
float  *block_clusters_d;											
/* per block calculation of deltas */
int    *block_deltas_d;												
=======
unsigned int num_threads_perdim = THREADS_PER_DIM;					/* sqrt(256) -- see references for this choice */
unsigned int num_blocks_perdim = BLOCKS_PER_DIM;					/* temporary */
unsigned int num_threads = num_threads_perdim*num_threads_perdim;	/* number of threads */
unsigned int num_blocks = num_blocks_perdim*num_blocks_perdim;		/* number of blocks */

/* _d denotes it resides on the device */
int    *membership_new;												/* newly assignment membership */
float  *feature_d;													/* inverted data array */
float  *feature_flipped_d;											/* original (not inverted) data array */
int    *membership_d;												/* membership on the device */
float  *block_new_centers;											/* sum of points in a cluster (per block) */
float  *clusters_d;													/* cluster centers on the device */
float  *block_clusters_d;											/* per block calculation of cluster centers */
int    *block_deltas_d;												/* per block calculation of deltas */
>>>>>>> ac67b0ac89e7e88a81b0ae127701263ca7241fd2

////////////////////////////////////////////////////////////////////////////////
void addBenchmarkSpecOptions(OptionParser &op) {
    op.addOption("maxClusters", OPT_INT, "5", "maximum number of clusters allowed");
    op.addOption("minClusters", OPT_INT, "5", "minimum number of clusters allowed");
    op.addOption("threshold", OPT_FLOAT, "0.001", "threshold value");
    op.addOption("loops", OPT_INT, "1", "iteration for each number of clusters");
    op.addOption("binaryInput", OPT_BOOL, "", "input file is in binary format");
    op.addOption("rmse", OPT_BOOL, "", "calculate RMSE (default off)");
    op.addOption("outputCenters", OPT_BOOL, "", "output cluster center coordinates (default off)");
}

void RunBenchmark(ResultDatabase &resultDB, OptionParser &op) {
    setup(resultDB, op);
}
////////////////////////////////////////////////////////////////////////////////

/* -------------- allocateMemory() ------------------- */
/* allocate device memory, calculate number of blocks and threads, and invert the data array */
void allocateMemory(int npoints, int nfeatures, int nclusters, float **features)
{	
	num_blocks = npoints / num_threads;
	if (npoints % num_threads > 0)		/* defeat truncation */
		num_blocks++;

	num_blocks_perdim = sqrt((double) num_blocks);
	while (num_blocks_perdim * num_blocks_perdim < num_blocks)	// defeat truncation (should run once)
		num_blocks_perdim++;

	num_blocks = num_blocks_perdim*num_blocks_perdim;

	/* allocate memory for memory_new[] and initialize to -1 (host) */
	membership_new = (int*) malloc(npoints * sizeof(int));
	for(int i=0;i<npoints;i++) {
		membership_new[i] = -1;
	}

	/* allocate memory for block_new_centers[] (host) */
	block_new_centers = (float *) malloc(nclusters*nfeatures*sizeof(float));
	
	/* allocate memory for feature_flipped_d[][], feature_d[][] (device) */
	hipMalloc((void**) &feature_flipped_d, npoints*nfeatures*sizeof(float));
	hipMemcpy(feature_flipped_d, features[0], npoints*nfeatures*sizeof(float), hipMemcpyHostToDevice);
	hipMalloc((void**) &feature_d, npoints*nfeatures*sizeof(float));
		
	/* invert the data array (kernel execution) */	
	invert_mapping<<<num_blocks,num_threads>>>(feature_flipped_d,feature_d,npoints,nfeatures);
<<<<<<< HEAD
    CHECK_CUDA_ERROR();
=======
>>>>>>> ac67b0ac89e7e88a81b0ae127701263ca7241fd2
		
	/* allocate memory for membership_d[] and clusters_d[][] (device) */
	hipMalloc((void**) &membership_d, npoints*sizeof(int));
	hipMalloc((void**) &clusters_d, nclusters*nfeatures*sizeof(float));

	
#ifdef BLOCK_DELTA_REDUCE
	// allocate array to hold the per block deltas on the gpu side
	
	hipMalloc((void**) &block_deltas_d, num_blocks_perdim * num_blocks_perdim * sizeof(int));
	//hipMemcpy(block_delta_d, &delta_h, sizeof(int), hipMemcpyHostToDevice);
#endif

#ifdef BLOCK_CENTER_REDUCE
	// allocate memory and copy to card cluster  array in which to accumulate center points for the next iteration
<<<<<<< HEAD
    hipMalloc((void**) &block_clusters_d, 
            num_blocks_perdim * num_blocks_perdim * 
            nclusters * nfeatures * sizeof(float));
=======
	hipMalloc((void**) &block_clusters_d, 
        num_blocks_perdim * num_blocks_perdim * 
        nclusters * nfeatures * sizeof(float));
>>>>>>> ac67b0ac89e7e88a81b0ae127701263ca7241fd2
	//hipMemcpy(new_clusters_d, new_centers[0], nclusters*nfeatures*sizeof(float), hipMemcpyHostToDevice);
#endif

}
/* -------------- allocateMemory() end ------------------- */

/* -------------- deallocateMemory() ------------------- */
/* free host and device memory */
void deallocateMemory()
{
	free(membership_new);
	free(block_new_centers);
	hipFree(feature_d);
	hipFree(feature_flipped_d);
	hipFree(membership_d);

	hipFree(clusters_d);
#ifdef BLOCK_CENTER_REDUCE
    hipFree(block_clusters_d);
#endif
#ifdef BLOCK_DELTA_REDUCE
    hipFree(block_deltas_d);
#endif
}
/* -------------- deallocateMemory() end ------------------- */


/* ------------------- kmeansCuda() ------------------------ */    
int	// delta -- had problems when return value was of float type
kmeansCuda(float  **feature,				/* in: [npoints][nfeatures] */
           int      nfeatures,				/* number of attributes for each point */
           int      npoints,				/* number of data points */
           int      nclusters,				/* number of clusters */
           int     *membership,				/* which cluster the point belongs to */
		   float  **clusters,				/* coordinates of cluster centers */
		   int     *new_centers_len,		/* number of elements in each cluster */
<<<<<<< HEAD
           float  **new_centers,			/* sum of elements in each cluster */
           double &transferTime,
           double &kernelTime,
		   ResultDatabase &resultDB)
=======
           float  **new_centers				/* sum of elements in each cluster */
		   )
>>>>>>> ac67b0ac89e7e88a81b0ae127701263ca7241fd2
{
	int delta = 0;			/* if point has moved */
	int i,j;				/* counters */

<<<<<<< HEAD
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsedTime;

    hipEventRecord(start, 0);
    /* copy membership (host to device) */
    hipMemcpy(membership_d, membership_new, npoints*sizeof(int), hipMemcpyHostToDevice);
    /* copy clusters (host to device) */
    hipMemcpy(clusters_d, clusters[0], nclusters*nfeatures*sizeof(float), hipMemcpyHostToDevice);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    transferTime += elapsedTime * 1.e-3; // convert to seconds

    hipError_t err;
=======

	hipSetDevice(1);

	/* copy membership (host to device) */
	hipMemcpy(membership_d, membership_new, npoints*sizeof(int), hipMemcpyHostToDevice);

	/* copy clusters (host to device) */
	hipMemcpy(clusters_d, clusters[0], nclusters*nfeatures*sizeof(float), hipMemcpyHostToDevice);
>>>>>>> ac67b0ac89e7e88a81b0ae127701263ca7241fd2

	/* set up texture */
    hipChannelFormatDesc chDesc0 = hipCreateChannelDesc<float>();
    t_features.filterMode = hipFilterModePoint;   
    t_features.normalized = false;
    t_features.channelDesc = chDesc0;

<<<<<<< HEAD
	err = hipBindTexture(NULL, &t_features, feature_d, &chDesc0, npoints*nfeatures*sizeof(float));
    if(err != hipSuccess) {
        printf("Error: Couldn't bind features array to texture, %d", err);
    }
=======
	hipBindTexture(NULL, &t_features, feature_d, &chDesc0, npoints*nfeatures*sizeof(float));
>>>>>>> ac67b0ac89e7e88a81b0ae127701263ca7241fd2

	hipChannelFormatDesc chDesc1 = hipCreateChannelDesc<float>();
    t_features_flipped.filterMode = hipFilterModePoint;   
    t_features_flipped.normalized = false;
    t_features_flipped.channelDesc = chDesc1;

<<<<<<< HEAD
	err = hipBindTexture(NULL, &t_features_flipped, feature_flipped_d, &chDesc1, npoints*nfeatures*sizeof(float));
    if(err != hipSuccess) {
        printf("Error: Couldn't bind features_flipped array to texture, %d", err);
    }
=======
	hipBindTexture(NULL, &t_features_flipped, feature_flipped_d, &chDesc1, npoints*nfeatures*sizeof(float));
>>>>>>> ac67b0ac89e7e88a81b0ae127701263ca7241fd2

	hipChannelFormatDesc chDesc2 = hipCreateChannelDesc<float>();
    t_clusters.filterMode = hipFilterModePoint;   
    t_clusters.normalized = false;
    t_clusters.channelDesc = chDesc2;

<<<<<<< HEAD
	err = hipBindTexture(NULL, &t_clusters, clusters_d, &chDesc2, nclusters*nfeatures*sizeof(float));
    if(err != hipSuccess) {
        printf("Error: Couldn't bind clusters array to texture, %d", err);
    }

  hipEventRecord(start, 0);
	/* copy clusters to constant memory */
	hipMemcpyToSymbol(HIP_SYMBOL("c_clusters"),clusters[0],nclusters*nfeatures*sizeof(float),0,hipMemcpyHostToDevice);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  transferTime += elapsedTime * 1.e-3; // convert to seconds
=======
	hipBindTexture(NULL, &t_clusters, clusters_d, &chDesc2, nclusters*nfeatures*sizeof(float));

	/* copy clusters to constant memory */
	hipMemcpyToSymbol(HIP_SYMBOL("c_clusters"),clusters[0],nclusters*nfeatures*sizeof(float),0,hipMemcpyHostToDevice);
>>>>>>> ac67b0ac89e7e88a81b0ae127701263ca7241fd2


    /* setup execution parameters.
	   changed to 2d (source code on NVIDIA CUDA Programming Guide) */
    dim3  grid( num_blocks_perdim, num_blocks_perdim );
    dim3  threads( num_threads_perdim*num_threads_perdim );
    
	/* execute the kernel */
<<<<<<< HEAD
    hipEventRecord(start, 0);
=======
>>>>>>> ac67b0ac89e7e88a81b0ae127701263ca7241fd2
    kmeansPoint<<< grid, threads >>>( feature_d,
                                      nfeatures,
                                      npoints,
                                      nclusters,
                                      membership_d,
                                      clusters_d,
									  block_clusters_d,
									  block_deltas_d);

	hipDeviceSynchronize();
<<<<<<< HEAD
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    kernelTime += elapsedTime * 1.e-3;
    //CHECK_CUDA_ERROR();

  hipEventRecord(start, 0);
	/* copy back membership (device to host) */
	hipMemcpy(membership_new, membership_d, npoints*sizeof(int), hipMemcpyDeviceToHost);	
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  transferTime += elapsedTime * 1.e-3; // convert to seconds
=======

	/* copy back membership (device to host) */
	hipMemcpy(membership_new, membership_d, npoints*sizeof(int), hipMemcpyDeviceToHost);	
>>>>>>> ac67b0ac89e7e88a81b0ae127701263ca7241fd2

#ifdef BLOCK_CENTER_REDUCE
    /*** Copy back arrays of per block sums ***/
    float * block_clusters_h = (float *) malloc(
        num_blocks_perdim * num_blocks_perdim * 
        nclusters * nfeatures * sizeof(float));
        
<<<<<<< HEAD
  hipEventRecord(start, 0);
=======
>>>>>>> ac67b0ac89e7e88a81b0ae127701263ca7241fd2
	hipMemcpy(block_clusters_h, block_clusters_d, 
        num_blocks_perdim * num_blocks_perdim * 
        nclusters * nfeatures * sizeof(float), 
        hipMemcpyDeviceToHost);
<<<<<<< HEAD
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  transferTime += elapsedTime * 1.e-3; // convert to seconds
=======
>>>>>>> ac67b0ac89e7e88a81b0ae127701263ca7241fd2
#endif
#ifdef BLOCK_DELTA_REDUCE
    int * block_deltas_h = (int *) malloc(
        num_blocks_perdim * num_blocks_perdim * sizeof(int));
        
<<<<<<< HEAD
  hipEventRecord(start, 0);
	hipMemcpy(block_deltas_h, block_deltas_d, 
        num_blocks_perdim * num_blocks_perdim * sizeof(int), 
        hipMemcpyDeviceToHost);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  transferTime += elapsedTime * 1.e-3; // convert to seconds
=======
	hipMemcpy(block_deltas_h, block_deltas_d, 
        num_blocks_perdim * num_blocks_perdim * sizeof(int), 
        hipMemcpyDeviceToHost);
>>>>>>> ac67b0ac89e7e88a81b0ae127701263ca7241fd2
#endif
    
	/* for each point, sum data points in each cluster
	   and see if membership has changed:
	     if so, increase delta and change old membership, and update new_centers;
	     otherwise, update new_centers */
	delta = 0;
	for (i = 0; i < npoints; i++)
	{		
		int cluster_id = membership_new[i];
		new_centers_len[cluster_id]++;
		if (membership_new[i] != membership[i])
		{
#ifdef CPU_DELTA_REDUCE
			delta++;
#endif
			membership[i] = membership_new[i];
		}
#ifdef CPU_CENTER_REDUCE
		for (j = 0; j < nfeatures; j++)
		{			
			new_centers[cluster_id][j] += feature[i][j];
		}
#endif
	}
	

#ifdef BLOCK_DELTA_REDUCE	
    /*** calculate global sums from per block sums for delta and the new centers ***/    
	
	//debug
	//printf("\t \t reducing %d block sums to global sum \n",num_blocks_perdim * num_blocks_perdim);
    for(i = 0; i < num_blocks_perdim * num_blocks_perdim; i++) {
		//printf("block %d delta is %d \n",i,block_deltas_h[i]);
        delta += block_deltas_h[i];
    }
        
#endif
#ifdef BLOCK_CENTER_REDUCE	
	
	for(int j = 0; j < nclusters;j++) {
		for(int k = 0; k < nfeatures;k++) {
			block_new_centers[j*nfeatures + k] = 0.f;
		}
	}

    for(i = 0; i < num_blocks_perdim * num_blocks_perdim; i++) {
		for(int j = 0; j < nclusters;j++) {
			for(int k = 0; k < nfeatures;k++) {
				block_new_centers[j*nfeatures + k] += block_clusters_h[i * nclusters*nfeatures + j * nfeatures + k];
			}
		}
    }
	

#ifdef CPU_CENTER_REDUCE
	//debug
	/*for(int j = 0; j < nclusters;j++) {
		for(int k = 0; k < nfeatures;k++) {
			if(new_centers[j][k] >	1.001 * block_new_centers[j*nfeatures + k] || new_centers[j][k] <	0.999 * block_new_centers[j*nfeatures + k]) {
				printf("\t \t for %d:%d, normal value is %e and gpu reduced value id %e \n",j,k,new_centers[j][k],block_new_centers[j*nfeatures + k]);
			}
		}
	}*/
#endif

#ifdef BLOCK_CENTER_REDUCE
	for(int j = 0; j < nclusters;j++) {
		for(int k = 0; k < nfeatures;k++)
			new_centers[j][k]= block_new_centers[j*nfeatures + k];		
	}
#endif

#endif

	return delta;
	
}
/* ------------------- kmeansCuda() end ------------------------ */    
