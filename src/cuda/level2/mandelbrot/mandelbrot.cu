#include "hip/hip_runtime.h"
/** @file histo-global.cu histogram with global memory atomics */

#include <png.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include "cudacommon.h"
#include "ResultDatabase.h"
#include "OptionParser.h"

/** block size along */
#define BSX 64
#define BSY 4
/** maximum recursion depth */
#define MAX_DEPTH 4
/** region below which do per-pixel */
#define MIN_SIZE 32
/** subdivision factor along each axis */
#define SUBDIV 4
/** subdivision when launched from host */
#define INIT_SUBDIV 32
/** binary operation for common dwell "reduction": MAX_DWELL + 1 = neutral
		element, -1 = dwells are different */
#define DIFF_DWELL (-1)

float kernelTime, transferTime;
hipEvent_t start, stop;
float elapsed;

/** a useful function to compute the number of threads */
__host__ __device__ int divup(int x, int y) { return x / y + (x % y ? 1 : 0); }

/** a simple complex type */
struct complex {
	__host__ __device__ complex(float re, float im = 0) {
		this->re = re;
		this->im = im;
	}
	/** real and imaginary part */
	float re, im;
}; // struct complex

// operator overloads for complex numbers
inline __host__ __device__ complex operator+
(const complex &a, const complex &b) {
	return complex(a.re + b.re, a.im + b.im);
}
inline __host__ __device__ complex operator-
(const complex &a) { return complex(-a.re, -a.im); }
inline __host__ __device__ complex operator-
(const complex &a, const complex &b) {
	return complex(a.re - b.re, a.im - b.im);
}
inline __host__ __device__ complex operator*
(const complex &a, const complex &b) {
	return complex(a.re * b.re - a.im * b.im, a.im * b.re + a.re * b.im);
}
inline __host__ __device__ float abs2(const complex &a) {
	return a.re * a.re + a.im * a.im;
}
inline __host__ __device__ complex operator/
(const complex &a, const complex &b) {
	float invabs2 = 1 / abs2(b);
	return complex((a.re * b.re + a.im * b.im) * invabs2,
								 (a.im * b.re - b.im * a.re) * invabs2);
}  // operator/
#define BS 256

/** computes the dwell for a single pixel */
__device__ int pixel_dwell
(int w, int h, complex cmin, complex cmax, int x, int y, int MAX_DWELL) {
	complex dc = cmax - cmin;
	float fx = (float)x / w, fy = (float)y / h;
	complex c = cmin + complex(fx * dc.re, fy * dc.im);
	int dwell = 0;
	complex z = c;
	while(dwell < MAX_DWELL && abs2(z) < 2 * 2) {
		z = z * z + c;
		dwell++;
	}
	return dwell;
}  // pixel_dwell

__device__ int same_dwell(int d1, int d2, int MAX_DWELL) {
    int NEUT_DWELL = MAX_DWELL + 1;
	if(d1 == d2)
		return d1;
	else if(d1 == NEUT_DWELL || d2 == NEUT_DWELL)
		return min(d1, d2);
	else
		return DIFF_DWELL;
}  // same_dwell

/** evaluates the common border dwell, if it exists */
__device__ int border_dwell
(int w, int h, complex cmin, complex cmax, int x0, int y0, int d, int MAX_DWELL) {
	// check whether all boundary pixels have the same dwell
	int tid = threadIdx.y * blockDim.x + threadIdx.x;
	int bs = blockDim.x * blockDim.y;
	int comm_dwell = MAX_DWELL + 1;
	// for all boundary pixels, distributed across threads
	for(int r = tid; r < d; r += bs) {
		// for each boundary: b = 0 is east, then counter-clockwise
		for(int b = 0; b < 4; b++) {
			int x = b % 2 != 0 ? x0 + r : (b == 0 ? x0 + d - 1 : x0);
			int y = b % 2 == 0 ? y0 + r : (b == 1 ? y0 + d - 1 : y0);
			int dwell = pixel_dwell(w, h, cmin, cmax, x, y, MAX_DWELL);
			comm_dwell = same_dwell(comm_dwell, dwell, MAX_DWELL);
		}
	}  // for all boundary pixels
	// reduce across threads in the block
	__shared__ int ldwells[BSX * BSY];
	int nt = min(d, BSX * BSY);
	if(tid < nt)
		ldwells[tid] = comm_dwell;
	__syncthreads();
	for(; nt > 1; nt /= 2) {
		if(tid < nt / 2)
			ldwells[tid] = same_dwell(ldwells[tid], ldwells[tid + nt / 2], MAX_DWELL);
		__syncthreads();
	}
	return ldwells[0];
}  // border_dwell

/** the kernel to fill the image region with a specific dwell value */
__global__ void dwell_fill_k
(int *dwells, int w, int x0, int y0, int d, int dwell) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	if(x < d && y < d) {
		x += x0, y += y0;
		dwells[y * w + x] = dwell;
	}
}  // dwell_fill_k

/** the kernel to fill in per-pixel values of the portion of the Mandelbrot set
		*/
__global__ void mandelbrot_pixel_k
(int *dwells, int w, int h, complex cmin, complex cmax, int x0, int y0, int d, int MAX_DWELL) {
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;
	if(x < d && y < d) {
		x += x0, y += y0;
		dwells[y * w + x] = pixel_dwell(w, h, cmin, cmax, x, y, MAX_DWELL);
	}
}  // mandelbrot_pixel_k

/** computes the dwells for Mandelbrot image 
		@param dwells the output array
		@param w the width of the output image
		@param h the height of the output image
		@param cmin the complex value associated with the left-bottom corner of the
		image
		@param cmax the complex value associated with the right-top corner of the
		image
 */
__global__ void mandelbrot_k
(int *dwells, int w, int h, complex cmin, complex cmax, int MAX_DWELL) {
	// complex value to start iteration (c)
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int dwell = pixel_dwell(w, h, cmin, cmax, x, y, MAX_DWELL);
	dwells[y * w + x] = dwell;
}  // mandelbrot_k

/** computes the dwells for Mandelbrot image using dynamic parallelism; one
		block is launched per pixel
		@param dwells the output array
		@param w the width of the output image
		@param h the height of the output image
		@param cmin the complex value associated with the left-bottom corner of the
		image
		@param cmax the complex value associated with the right-top corner of the
		image
		@param x0 the starting x coordinate of the portion to compute
		@param y0 the starting y coordinate of the portion to compute
		@param d the size of the portion to compute (the portion is always a square)
		@param depth kernel invocation depth
		@remarks the algorithm reverts to per-pixel Mandelbrot evaluation once
		either maximum depth or minimum size is reached
 */
__global__ void mandelbrot_block_k
(int *dwells, int w, int h, complex cmin, complex cmax, int x0, int y0, 
 int d, int depth, int MAX_DWELL) {
	x0 += d * blockIdx.x, y0 += d * blockIdx.y;
	int comm_dwell = border_dwell(w, h, cmin, cmax, x0, y0, d, MAX_DWELL);
	if(threadIdx.x == 0 && threadIdx.y == 0) {
		if(comm_dwell != DIFF_DWELL) {
			// uniform dwell, just fill
			dim3 bs(BSX, BSY), grid(divup(d, BSX), divup(d, BSY));
			dwell_fill_k<<<grid, bs>>>(dwells, w, x0, y0, d, comm_dwell);
		} else if(depth + 1 < MAX_DEPTH && d / SUBDIV > MIN_SIZE) {
			// subdivide recursively
			dim3 bs(blockDim.x, blockDim.y), grid(SUBDIV, SUBDIV);
			mandelbrot_block_k<<<grid, bs>>>
				(dwells, w, h, cmin, cmax, x0, y0, d / SUBDIV, depth + 1, MAX_DWELL);
		} else {
			// leaf, per-pixel kernel
			dim3 bs(BSX, BSY), grid(divup(d, BSX), divup(d, BSY));
			mandelbrot_pixel_k<<<grid, bs>>>
				(dwells, w, h, cmin, cmax, x0, y0, d, MAX_DWELL);
		}
		//CUDA_SAFE_CALL(hipGetLastError());
	}
}  // mandelbrot_block_k

void mandelbrot(int size, int MAX_DWELL) {
	// allocate memory
	int w = size, h = size;
	size_t dwell_sz = w * h * sizeof(int);
	int *h_dwells, *d_dwells;
	CUDA_SAFE_CALL(hipMalloc((void**)&d_dwells, dwell_sz));
	h_dwells = (int*)malloc(dwell_sz);

	// compute the dwells, copy them back
	dim3 bs(64, 4), grid(divup(w, bs.x), divup(h, bs.y));
    hipEventRecord(start, 0);
	mandelbrot_k<<<grid, bs>>>
		(d_dwells, w, h, complex(-1.5, -1), complex(0.5, 1), MAX_DWELL);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    kernelTime += elapsed * 1.e-3;

    CHECK_CUDA_ERROR();
	CUDA_SAFE_CALL(hipDeviceSynchronize());
    hipEventRecord(start, 0);
	CUDA_SAFE_CALL(hipMemcpy(h_dwells, d_dwells, dwell_sz, hipMemcpyDeviceToHost));
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    transferTime += elapsed * 1.e-3;

	// free data
	hipFree(d_dwells);
	free(h_dwells);
}

void mandelbrot_dyn(int size, int MAX_DWELL) {
	// allocate memory
	int w = size, h = size;
	size_t dwell_sz = w * h * sizeof(int);
	int *h_dwells, *d_dwells;
	CUDA_SAFE_CALL(hipMalloc((void**)&d_dwells, dwell_sz));
	h_dwells = (int*)malloc(dwell_sz);

	// compute the dwells, copy them back
	dim3 bs(BSX, BSY), grid(INIT_SUBDIV, INIT_SUBDIV);
    hipEventRecord(start, 0);
	mandelbrot_block_k<<<grid, bs>>>
		(d_dwells, w, h, complex(-1.5, -1), complex(0.5, 1), 0, 0, w / INIT_SUBDIV, 1, MAX_DWELL);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    kernelTime += elapsed * 1.e-3;

    CHECK_CUDA_ERROR();
	CUDA_SAFE_CALL(hipDeviceSynchronize());
    hipEventRecord(start, 0);
	CUDA_SAFE_CALL(hipMemcpy(h_dwells, d_dwells, dwell_sz, hipMemcpyDeviceToHost));
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    transferTime += elapsed * 1.e-3;

	// free data
	hipFree(d_dwells);
	free(h_dwells);
}

void addBenchmarkSpecOptions(OptionParser &op) {
    op.addOption("imageSize", OPT_INT, "0", "image height and width");
    op.addOption("iterations", OPT_INT, "0", "iterations of algorithm (the more iterations, the greater speedup from dynamic parallelism)");
}

void RunBenchmark(ResultDatabase &resultDB, OptionParser &op) {

    hipEventCreate(&start);
    hipEventCreate(&stop);

    int imageSize = op.getOptionInt("imageSize");
    int iters = op.getOptionInt("iterations");
    if(imageSize == 0 || iters == 0) {
        int imageSizes[4] = {2 << 11, 2 << 13, 2 << 14, 2 << 15};
        int iterSizes[4] = {32, 128, 512, 1024};
        imageSize = imageSizes[op.getOptionInt("size") - 1];
        iters = iterSizes[op.getOptionInt("size") - 1];
    }
    
    printf("Image Size: %d by %d\n", imageSize, imageSize);
    printf("Num Iterations: %d\n", iters);
#ifdef DYNAMIC_PARALLELISM
    printf("Using dynamic parallelism\n");
#else
    printf("Not using dynamic parallelism\n");
#endif

    int passes = op.getOptionInt("passes");
    for(int i = 0; i < passes; i++) {
        kernelTime = 0.0f;
        transferTime = 0.0f;
        printf("Pass %d:\n", i);
#ifdef DYNAMIC_PARALLELISM
        mandelbrot_dyn(imageSize, iters);
#else
        mandelbrot(imageSize, iters);
#endif
        printf("Done.\n");
        char atts[1024];
        sprintf(atts, "img:%d,iter:%d", imageSize, iters);
        resultDB.AddResult("mandelbrot_kernel_time", atts, "sec", kernelTime);
        resultDB.AddResult("mandelbrot_transfer_time", atts, "sec", transferTime);
        resultDB.AddResult("mandelbrot_parity", atts, "N", transferTime / kernelTime);
    }

}
