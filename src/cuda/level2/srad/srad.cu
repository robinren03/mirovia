#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include "srad.h"

#include "cudacommon.h"
#include "ResultDatabase.h"
#include "OptionParser.h"

// includes, project
#include <hip/hip_runtime.h>

// includes, kernels
//#include "srad_kernel.cu"

float kernelTime = 0.0f;
float transferTime = 0.0f;
hipEvent_t start, stop;
float elapsed;

void random_matrix(float *I, int rows, int cols);
void runTest( int argc, char** argv);
void srad(ResultDatabase &resultDB, int imageSize, int speckleSize, int iters);

__global__ void
srad_cuda_1(
		  float *E_C, 
		  float *W_C, 
		  float *N_C, 
		  float *S_C,
		  float * J_cuda, 
		  float * C_cuda, 
		  int cols, 
		  int rows, 
		  float q0sqr
) 
{

  //block id
  int bx = blockIdx.x;
  int by = blockIdx.y;

  //thread id
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  
  //indices
  int index   = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + tx;
  int index_n = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + tx - cols;
  int index_s = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * BLOCK_SIZE + tx;
  int index_w = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty - 1;
  int index_e = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + BLOCK_SIZE;

  if(index_n >= rows * cols ||
     index_s >= rows * cols ||
     index_e >= rows * cols ||
     index_w >= rows * cols ||
     index_n < 0 ||
     index_s < 0||
     index_e < 0 ||
     index_w < 0) {
      return;
  }

  float n, w, e, s, jc, g2, l, num, den, qsqr, c;

  //shared memory allocation
  __shared__ float temp[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float temp_result[BLOCK_SIZE][BLOCK_SIZE];

  __shared__ float north[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float south[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float  east[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float  west[BLOCK_SIZE][BLOCK_SIZE];

  //load data to shared memory
  north[ty][tx] = J_cuda[index_n]; 
  south[ty][tx] = J_cuda[index_s];
  if ( by == 0 ){
  north[ty][tx] = J_cuda[BLOCK_SIZE * bx + tx]; 
  }
  else if ( by == gridDim.y - 1 ){
  south[ty][tx] = J_cuda[cols * BLOCK_SIZE * (gridDim.y - 1) + BLOCK_SIZE * bx + cols * ( BLOCK_SIZE - 1 ) + tx];
  }
   __syncthreads();
 
  west[ty][tx] = J_cuda[index_w];
  east[ty][tx] = J_cuda[index_e];

  if ( bx == 0 ){
  west[ty][tx] = J_cuda[cols * BLOCK_SIZE * by + cols * ty]; 
  }
  else if ( bx == gridDim.x - 1 ){
  east[ty][tx] = J_cuda[cols * BLOCK_SIZE * by + BLOCK_SIZE * ( gridDim.x - 1) + cols * ty + BLOCK_SIZE-1];
  }
 
  __syncthreads();
  
 

  temp[ty][tx]      = J_cuda[index];

  __syncthreads();

   jc = temp[ty][tx];

   if ( ty == 0 && tx == 0 ){ //nw
	n  = north[ty][tx] - jc;
    s  = temp[ty+1][tx] - jc;
    w  = west[ty][tx]  - jc; 
    e  = temp[ty][tx+1] - jc;
   }	    
   else if ( ty == 0 && tx == BLOCK_SIZE-1 ){ //ne
	n  = north[ty][tx] - jc;
    s  = temp[ty+1][tx] - jc;
    w  = temp[ty][tx-1] - jc; 
    e  = east[ty][tx] - jc;
   }
   else if ( ty == BLOCK_SIZE -1 && tx == BLOCK_SIZE - 1){ //se
	n  = temp[ty-1][tx] - jc;
    s  = south[ty][tx] - jc;
    w  = temp[ty][tx-1] - jc; 
    e  = east[ty][tx]  - jc;
   }
   else if ( ty == BLOCK_SIZE -1 && tx == 0 ){//sw
	n  = temp[ty-1][tx] - jc;
    s  = south[ty][tx] - jc;
    w  = west[ty][tx]  - jc; 
    e  = temp[ty][tx+1] - jc;
   }

   else if ( ty == 0 ){ //n
	n  = north[ty][tx] - jc;
    s  = temp[ty+1][tx] - jc;
    w  = temp[ty][tx-1] - jc; 
    e  = temp[ty][tx+1] - jc;
   }
   else if ( tx == BLOCK_SIZE -1 ){ //e
	n  = temp[ty-1][tx] - jc;
    s  = temp[ty+1][tx] - jc;
    w  = temp[ty][tx-1] - jc; 
    e  = east[ty][tx] - jc;
   }
   else if ( ty == BLOCK_SIZE -1){ //s
	n  = temp[ty-1][tx] - jc;
    s  = south[ty][tx] - jc;
    w  = temp[ty][tx-1] - jc; 
    e  = temp[ty][tx+1] - jc;
   }
   else if ( tx == 0 ){ //w
	n  = temp[ty-1][tx] - jc;
    s  = temp[ty+1][tx] - jc;
    w  = west[ty][tx] - jc; 
    e  = temp[ty][tx+1] - jc;
   }
   else{  //the data elements which are not on the borders 
	n  = temp[ty-1][tx] - jc;
    s  = temp[ty+1][tx] - jc;
    w  = temp[ty][tx-1] - jc; 
    e  = temp[ty][tx+1] - jc;
   }


    g2 = ( n * n + s * s + w * w + e * e ) / (jc * jc);

    l = ( n + s + w + e ) / jc;

	num  = (0.5*g2) - ((1.0/16.0)*(l*l)) ;
	den  = 1 + (.25*l);
	qsqr = num/(den*den);

	// diffusion coefficent (equ 33)
	den = (qsqr-q0sqr) / (q0sqr * (1+q0sqr)) ;
	c = 1.0 / (1.0+den) ;

    // saturate diffusion coefficent
	if (c < 0){temp_result[ty][tx] = 0;}
	else if (c > 1) {temp_result[ty][tx] = 1;}
	else {temp_result[ty][tx] = c;}

    __syncthreads();

    C_cuda[index] = temp_result[ty][tx];
	E_C[index] = e;
	W_C[index] = w;
	S_C[index] = s;
	N_C[index] = n;

}

__global__ void
srad_cuda_2(
		  float *E_C, 
		  float *W_C, 
		  float *N_C, 
		  float *S_C,	
		  float * J_cuda, 
		  float * C_cuda, 
		  int cols, 
		  int rows, 
		  float lambda,
		  float q0sqr
) 
{
	//block id
	int bx = blockIdx.x;
    int by = blockIdx.y;

	//thread id
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	//indices
    int index   = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + tx;
	int index_s = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * BLOCK_SIZE + tx;
    int index_e = cols * BLOCK_SIZE * by + BLOCK_SIZE * bx + cols * ty + BLOCK_SIZE;
	float cc, cn, cs, ce, cw, d_sum;

	//shared memory allocation
	__shared__ float south_c[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float  east_c[BLOCK_SIZE][BLOCK_SIZE];

    __shared__ float c_cuda_temp[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float c_cuda_result[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float temp[BLOCK_SIZE][BLOCK_SIZE];

    //load data to shared memory
	temp[ty][tx]      = J_cuda[index];

    __syncthreads();
	 
	south_c[ty][tx] = C_cuda[index_s];

	if ( by == gridDim.y - 1 ){
	south_c[ty][tx] = C_cuda[cols * BLOCK_SIZE * (gridDim.y - 1) + BLOCK_SIZE * bx + cols * ( BLOCK_SIZE - 1 ) + tx];
	}
	__syncthreads();
	 
	 
	east_c[ty][tx] = C_cuda[index_e];
	
	if ( bx == gridDim.x - 1 ){
	east_c[ty][tx] = C_cuda[cols * BLOCK_SIZE * by + BLOCK_SIZE * ( gridDim.x - 1) + cols * ty + BLOCK_SIZE-1];
	}
	 
    __syncthreads();
  
    c_cuda_temp[ty][tx]      = C_cuda[index];

    __syncthreads();

	cc = c_cuda_temp[ty][tx];

   if ( ty == BLOCK_SIZE -1 && tx == BLOCK_SIZE - 1){ //se
	cn  = cc;
    cs  = south_c[ty][tx];
    cw  = cc; 
    ce  = east_c[ty][tx];
   } 
   else if ( tx == BLOCK_SIZE -1 ){ //e
	cn  = cc;
    cs  = c_cuda_temp[ty+1][tx];
    cw  = cc; 
    ce  = east_c[ty][tx];
   }
   else if ( ty == BLOCK_SIZE -1){ //s
	cn  = cc;
    cs  = south_c[ty][tx];
    cw  = cc; 
    ce  = c_cuda_temp[ty][tx+1];
   }
   else{ //the data elements which are not on the borders 
	cn  = cc;
    cs  = c_cuda_temp[ty+1][tx];
    cw  = cc; 
    ce  = c_cuda_temp[ty][tx+1];
   }

   // divergence (equ 58)
   d_sum = cn * N_C[index] + cs * S_C[index] + cw * W_C[index] + ce * E_C[index];

   // image update (equ 61)
   c_cuda_result[ty][tx] = temp[ty][tx] + 0.25 * lambda * d_sum;

   __syncthreads();
              
   J_cuda[index] = c_cuda_result[ty][tx];
    
}

void addBenchmarkSpecOptions(OptionParser &op) {
    op.addOption("imageSize", OPT_INT, "0", "image height and width");
    op.addOption("speckleSize", OPT_INT, "0", "speckle height and width");
    op.addOption("iterations", OPT_INT, "0", "iterations of algorithm");
}

void RunBenchmark(ResultDatabase &resultDB, OptionParser &op) 
{
  printf("WG size of kernel = %d X %d\n", BLOCK_SIZE, BLOCK_SIZE);
  int imageSize = op.getOptionInt("imageSize");
  int speckleSize = op.getOptionInt("speckleSize");
  int iters = op.getOptionInt("iterations");
  if(imageSize == 0 || speckleSize == 0 || iters == 0) {
      int imageSizes[4] = {128, 512, 4096, 8192};
      int iterSizes[4] = {5, 50, 100, 200};
      imageSize = imageSizes[op.getOptionInt("size") - 1];
      speckleSize = imageSize / 2;
      iters = iterSizes[op.getOptionInt("size") - 1];
  }
  hipEventCreate(&start);
  hipEventCreate(&stop);

  printf("Image Size: %d by %d\n", imageSize, imageSize);
  printf("Speckle size: %d by %d\n", speckleSize, speckleSize);
  printf("Num Iterations: %d\n", iters);

  int passes = op.getOptionInt("passes");
  for(int i = 0; i < passes; i++) {
      kernelTime = 0.0f;
      transferTime = 0.0f;
      printf("Pass %d:\n", i);
      srad(resultDB, imageSize, speckleSize, iters);
      printf("Done.\n");
      char atts[1024];
      sprintf(atts, "img:%d,speckle:%d,iter:%d", imageSize, speckleSize, iters);
      resultDB.AddResult("srad_kernel_time", atts, "sec", kernelTime);
      resultDB.AddResult("srad_transfer_time", atts, "sec", transferTime);
      resultDB.AddResult("srad_parity", atts, "N", transferTime / kernelTime);
  }
}


void srad(ResultDatabase &resultDB, int imageSize, int speckleSize, int iters) 
{
    int rows, cols, size_I, size_R, niter = 10, iter;
    float *I, *J, lambda, q0sqr, sum, sum2, tmp, meanROI,varROI ;

#ifdef CPU
	float Jc, G2, L, num, den, qsqr;
	int *iN,*iS,*jE,*jW, k;
	float *dN,*dS,*dW,*dE;
	float cN,cS,cW,cE,D;
#endif

#ifdef GPU
	
	float *J_cuda;
    float *C_cuda;
	float *E_C, *W_C, *N_C, *S_C;

#endif

	unsigned int r1, r2, c1, c2;
	float *c;
    
	
 
    rows = imageSize;  //number of rows in the domain
    cols = imageSize;  //number of cols in the domain
    if ((rows%16!=0) || (cols%16!=0)){
        fprintf(stderr, "rows and cols must be multiples of 16\n");
        exit(1);
    }
    r1   = 0;  //y1 position of the speckle
    r2   = speckleSize;  //y2 position of the speckle
    c1   = 0;  //x1 position of the speckle
    c2   = speckleSize;  //x2 position of the speckle
    lambda = 0.5; //Lambda value
    niter = iters; //number of iterations


	size_I = cols * rows;
    size_R = (r2-r1+1)*(c2-c1+1);   

	I = (float *)malloc( size_I * sizeof(float) );
    J = (float *)malloc( size_I * sizeof(float) );
	c  = (float *)malloc(sizeof(float)* size_I) ;


#ifdef CPU

    iN = (int *)malloc(sizeof(unsigned int*) * rows) ;
    iS = (int *)malloc(sizeof(unsigned int*) * rows) ;
    jW = (int *)malloc(sizeof(unsigned int*) * cols) ;
    jE = (int *)malloc(sizeof(unsigned int*) * cols) ;    


	dN = (float *)malloc(sizeof(float)* size_I) ;
    dS = (float *)malloc(sizeof(float)* size_I) ;
    dW = (float *)malloc(sizeof(float)* size_I) ;
    dE = (float *)malloc(sizeof(float)* size_I) ;    
    

    for (int i=0; i< rows; i++) {
        iN[i] = i-1;
        iS[i] = i+1;
    }    
    for (int j=0; j< cols; j++) {
        jW[j] = j-1;
        jE[j] = j+1;
    }
    iN[0]    = 0;
    iS[rows-1] = rows-1;
    jW[0]    = 0;
    jE[cols-1] = cols-1;

#endif

#ifdef GPU

	//Allocate device memory
    CUDA_SAFE_CALL(hipMalloc((void**)& J_cuda, sizeof(float)* size_I));
    CUDA_SAFE_CALL(hipMalloc((void**)& C_cuda, sizeof(float)* size_I));
	CUDA_SAFE_CALL(hipMalloc((void**)& E_C, sizeof(float)* size_I));
	CUDA_SAFE_CALL(hipMalloc((void**)& W_C, sizeof(float)* size_I));
	CUDA_SAFE_CALL(hipMalloc((void**)& S_C, sizeof(float)* size_I));
	CUDA_SAFE_CALL(hipMalloc((void**)& N_C, sizeof(float)* size_I));

	
#endif 

	//Generate a random matrix
	random_matrix(I, rows, cols);

    for (int k = 0;  k < size_I; k++ ) {
     	J[k] = (float)exp(I[k]) ;
    }
 for (iter=0; iter< niter; iter++){     
		sum=0; sum2=0;
        for (int i=r1; i<=r2; i++) {
            for (int j=c1; j<=c2; j++) {
                tmp   = J[i * cols + j];
                sum  += tmp ;
                sum2 += tmp*tmp;
            }
        }
        meanROI = sum / size_R;
        varROI  = (sum2 / size_R) - meanROI*meanROI;
        q0sqr   = varROI / (meanROI*meanROI);

#ifdef CPU
        
		for (int i = 0 ; i < rows ; i++) {
            for (int j = 0; j < cols; j++) { 
		
				k = i * cols + j;
				Jc = J[k];
 
				// directional derivates
                dN[k] = J[iN[i] * cols + j] - Jc;
                dS[k] = J[iS[i] * cols + j] - Jc;
                dW[k] = J[i * cols + jW[j]] - Jc;
                dE[k] = J[i * cols + jE[j]] - Jc;
			
                G2 = (dN[k]*dN[k] + dS[k]*dS[k] 
                    + dW[k]*dW[k] + dE[k]*dE[k]) / (Jc*Jc);

   		        L = (dN[k] + dS[k] + dW[k] + dE[k]) / Jc;

				num  = (0.5*G2) - ((1.0/16.0)*(L*L)) ;
                den  = 1 + (.25*L);
                qsqr = num/(den*den);
 
                // diffusion coefficent (equ 33)
                den = (qsqr-q0sqr) / (q0sqr * (1+q0sqr)) ;
                c[k] = 1.0 / (1.0+den) ;
                
                // saturate diffusion coefficent
                if (c[k] < 0) {c[k] = 0;}
                else if (c[k] > 1) {c[k] = 1;}
		}
	}
         for (int i = 0; i < rows; i++) {
            for (int j = 0; j < cols; j++) {        

                // current index
                k = i * cols + j;
                
                // diffusion coefficent
					cN = c[k];
					cS = c[iS[i] * cols + j];
					cW = c[k];
					cE = c[i * cols + jE[j]];

                // divergence (equ 58)
                D = cN * dN[k] + cS * dS[k] + cW * dW[k] + cE * dE[k];
                
                // image update (equ 61)
                J[k] = J[k] + 0.25*lambda*D;
            }
	}

#endif // CPU


#ifdef GPU

	//Currently the input size must be divided by 16 - the block size
	int block_x = cols/BLOCK_SIZE ;
    int block_y = rows/BLOCK_SIZE ;

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(block_x , block_y);
    

	//Copy data from main memory to device memory
    hipEventRecord(start, 0);
	CUDA_SAFE_CALL(hipMemcpy(J_cuda, J, sizeof(float) * size_I, hipMemcpyHostToDevice));
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    transferTime += elapsed * 1.e-3;

	//Run kernels
    hipEventRecord(start, 0);
	srad_cuda_1<<<dimGrid, dimBlock>>>(E_C, W_C, N_C, S_C, J_cuda, C_cuda, cols, rows, q0sqr); 
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    kernelTime += elapsed * 1.e-3;
    CHECK_CUDA_ERROR();

    hipEventRecord(start, 0);
	srad_cuda_2<<<dimGrid, dimBlock>>>(E_C, W_C, N_C, S_C, J_cuda, C_cuda, cols, rows, lambda, q0sqr); 
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    kernelTime += elapsed * 1.e-3;
    CHECK_CUDA_ERROR();

	//Copy data from device memory to main memory
    hipEventRecord(start, 0);
    CUDA_SAFE_CALL(hipMemcpy(J, J_cuda, sizeof(float) * size_I, hipMemcpyDeviceToHost));
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    transferTime += elapsed * 1.e-3;

#endif   
}

    hipDeviceSynchronize();

#ifdef OUTPUT
    //Printing output	
		printf("Printing Output:\n"); 
    for( int i = 0 ; i < rows ; i++){
		for ( int j = 0 ; j < cols ; j++){
         printf("%.5f ", J[i * cols + j]); 
		}	
     printf("\n"); 
   }
#endif 


	free(I);
	free(J);
#ifdef CPU
	free(iN); free(iS); free(jW); free(jE);
    free(dN); free(dS); free(dW); free(dE);
#endif
#ifdef GPU
    CUDA_SAFE_CALL(hipFree(C_cuda));
	CUDA_SAFE_CALL(hipFree(J_cuda));
	CUDA_SAFE_CALL(hipFree(E_C));
	CUDA_SAFE_CALL(hipFree(W_C));
	CUDA_SAFE_CALL(hipFree(N_C));
	CUDA_SAFE_CALL(hipFree(S_C));
#endif 
	free(c);
  
}


void random_matrix(float *I, int rows, int cols){
    
	srand(7);
	
	for( int i = 0 ; i < rows ; i++){
		for ( int j = 0 ; j < cols ; j++){
		 I[i * cols + j] = rand()/(float)RAND_MAX ;
		}
	}

}
