#include "hip/hip_runtime.h"
// includes, system
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "srad.h"

#include "OptionParser.h"
#include "ResultDatabase.h"
#include "cudacommon.h"

// includes, project
#include <hip/hip_runtime.h>

// includes, kernels
#include "srad_kernel.cu"

#define SEED 7

float kernelTime = 0.0f;
float transferTime = 0.0f;
hipEvent_t start, stop;
float elapsed;
float *check;

void random_matrix(float *I, int rows, int cols);
void runTest(int argc, char **argv);
float srad(ResultDatabase &resultDB, OptionParser &op, float* matrix, int imageSize, int speckleSize, int iters);
float srad_gridsync(ResultDatabase &resultDB, OptionParser &op, float* matrix, int imageSize, int speckleSize, int iters);

void addBenchmarkSpecOptions(OptionParser &op) {
  op.addOption("imageSize", OPT_INT, "0", "image height and width");
  op.addOption("speckleSize", OPT_INT, "0", "speckle height and width");
  op.addOption("iterations", OPT_INT, "0", "iterations of algorithm");
  op.addOption("resultfile", OPT_STRING, "", "file to write results to");
}

void RunBenchmark(ResultDatabase &resultDB, OptionParser &op) {
  srand(SEED);
  printf("WG size of kernel = %d X %d\n", BLOCK_SIZE, BLOCK_SIZE);

  // set parameters
  int imageSize = op.getOptionInt("imageSize");
  int speckleSize = op.getOptionInt("speckleSize");
  int iters = op.getOptionInt("iterations");
  if (imageSize == 0 || speckleSize == 0 || iters == 0) {
    int imageSizes[4] = {128, 512, 4096, 2 << 13};
    int iterSizes[4] = {5, 10, 15, 20};
    imageSize = imageSizes[op.getOptionInt("size") - 1];
    speckleSize = imageSize / 2;
    iters = iterSizes[op.getOptionInt("size") - 1];
  }

  // create timing events
  hipEventCreate(&start);
  hipEventCreate(&stop);

  printf("Image Size: %d x %d\n", imageSize, imageSize);
  printf("Speckle size: %d x %d\n", speckleSize, speckleSize);
  printf("Num Iterations: %d\n", iters);

  // run workload
  int passes = op.getOptionInt("passes");
  for (int i = 0; i < passes; i++) {
    float *matrix = (float*)malloc(imageSize * imageSize * sizeof(float));
    random_matrix(matrix, imageSize, imageSize);
    printf("Pass %d:\n", i);
    float time = srad(resultDB, op, matrix, imageSize, speckleSize, iters);
    printf("Running SRAD...Done.\n");
#ifdef GRID_SYNC
    // if using cooperative groups, add result to compare the 2 times
    char atts[1024];
    sprintf(atts, "img:%d,speckle:%d,iter:%d", imageSize, speckleSize, iters);
    float time_gridsync = srad_gridsync(resultDB, op, matrix, imageSize, speckleSize, iters);
    if(time_gridsync == FLT_MAX) {
        printf("Running SRAD with cooperative groups...Failed.\n");
    } else {
        printf("Running SRAD with cooperative groups...Done.\n");
        resultDB.AddResult("srad_time/srad_cg_time", atts, "N", time/time_gridsync);
    }
#endif
      free(matrix);
  }
}

float srad(ResultDatabase &resultDB, OptionParser &op, float* matrix, int imageSize,
          int speckleSize, int iters) {
    kernelTime = 0.0f;
    transferTime = 0.0f;
    int rows, cols, size_I, size_R, niter, iter;
    float *I, *J, lambda, q0sqr, sum, sum2, tmp, meanROI, varROI;

#ifdef GPU

  float *J_cuda;
  float *C_cuda;
  float *E_C, *W_C, *N_C, *S_C;

#endif

  unsigned int r1, r2, c1, c2;
  float *c;

  rows = imageSize;  // number of rows in the domain
  cols = imageSize;  // number of cols in the domain
  if ((rows % 16 != 0) || (cols % 16 != 0)) {
    fprintf(stderr, "rows and cols must be multiples of 16\n");
    exit(1);
  }
  r1 = 0;            // y1 position of the speckle
  r2 = speckleSize;  // y2 position of the speckle
  c1 = 0;            // x1 position of the speckle
  c2 = speckleSize;  // x2 position of the speckle
  lambda = 0.5;      // Lambda value
  niter = iters;     // number of iterations

  size_I = cols * rows;
  size_R = (r2 - r1 + 1) * (c2 - c1 + 1);

  I = (float *)malloc(size_I * sizeof(float));
  J = (float *)malloc(size_I * sizeof(float));
  c = (float *)malloc(sizeof(float) * size_I);

#ifdef GPU

  // Allocate device memory
  CUDA_SAFE_CALL(hipMalloc((void **)&J_cuda, sizeof(float) * size_I));
  CUDA_SAFE_CALL(hipMalloc((void **)&C_cuda, sizeof(float) * size_I));
  CUDA_SAFE_CALL(hipMalloc((void **)&E_C, sizeof(float) * size_I));
  CUDA_SAFE_CALL(hipMalloc((void **)&W_C, sizeof(float) * size_I));
  CUDA_SAFE_CALL(hipMalloc((void **)&S_C, sizeof(float) * size_I));
  CUDA_SAFE_CALL(hipMalloc((void **)&N_C, sizeof(float) * size_I));

#endif

  // copy random matrix
  memcpy(I, matrix, rows*cols*sizeof(float));

  for (int k = 0; k < size_I; k++) {
    J[k] = (float)exp(I[k]);
  }
  for (iter = 0; iter < niter; iter++) {
    sum = 0;
    sum2 = 0;
    for (int i = r1; i <= r2; i++) {
      for (int j = c1; j <= c2; j++) {
        tmp = J[i * cols + j];
        sum += tmp;
        sum2 += tmp * tmp;
      }
    }
    meanROI = sum / size_R;
    varROI = (sum2 / size_R) - meanROI * meanROI;
    q0sqr = varROI / (meanROI * meanROI);

#ifdef GPU
    // Currently the input size must be divided by 16 - the block size
    int block_x = cols / BLOCK_SIZE;
    int block_y = rows / BLOCK_SIZE;

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(block_x, block_y);

    // Copy data from main memory to device memory
    hipEventRecord(start, 0);
    CUDA_SAFE_CALL(
        hipMemcpy(J_cuda, J, sizeof(float) * size_I, hipMemcpyHostToDevice));
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    transferTime += elapsed * 1.e-3;

    // Run kernels
    hipEventRecord(start, 0);
    srad_cuda_1<<<dimGrid, dimBlock>>>(E_C, W_C, N_C, S_C, J_cuda, C_cuda, cols,
                                       rows, q0sqr);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    kernelTime += elapsed * 1.e-3;
    CHECK_CUDA_ERROR();

    hipEventRecord(start, 0);
    srad_cuda_2<<<dimGrid, dimBlock>>>(E_C, W_C, N_C, S_C, J_cuda, C_cuda, cols,
                                       rows, lambda, q0sqr);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    kernelTime += elapsed * 1.e-3;
    CHECK_CUDA_ERROR();

    // Copy data from device memory to main memory
    hipEventRecord(start, 0);
    CUDA_SAFE_CALL(
        hipMemcpy(J, J_cuda, sizeof(float) * size_I, hipMemcpyDeviceToHost));
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    transferTime += elapsed * 1.e-3;
#endif
  }

    char atts[1024];
    sprintf(atts, "img:%d,speckle:%d,iter:%d", imageSize, speckleSize, iters);
    resultDB.AddResult("srad_kernel_time", atts, "sec", kernelTime);
    resultDB.AddResult("srad_transfer_time", atts, "sec", transferTime);
    resultDB.AddResult("srad_total_time", atts, "sec", kernelTime + transferTime);
    resultDB.AddResult("srad_parity", atts, "N", transferTime / kernelTime);

  string resultfile = op.getOptionString("resultfile");
  if(!resultfile.empty()) {
      // Printing output
      printf("Writing output to %s\n", resultfile.c_str());
      FILE *fp = NULL;
      fp = fopen(resultfile.c_str(), "w");
      if(!fp) {
          printf("Error: Unable to write to file %s\n", resultfile.c_str());
      } else {
          for (int i = 0; i < rows; i++) {
              for (int j = 0; j < cols; j++) {
                  fprintf(fp, "%.5f ", J[i * cols + j]);
              }
              fprintf(fp, "\n");
          }
          fclose(fp);
      }
  }
  // write results to validate with srad_gridsync
  check = (float*) malloc(sizeof(float) * size_I);
  for (int i = 0; i < rows; i++) {
      for (int j = 0; j < cols; j++) {
          check[i*cols+j] = J[i*cols+j];
      }
  }

  free(I);
  free(J);
  free(c);
#ifdef GPU
  CUDA_SAFE_CALL(hipFree(C_cuda));
  CUDA_SAFE_CALL(hipFree(J_cuda));
  CUDA_SAFE_CALL(hipFree(E_C));
  CUDA_SAFE_CALL(hipFree(W_C));
  CUDA_SAFE_CALL(hipFree(N_C));
  CUDA_SAFE_CALL(hipFree(S_C));
#endif
    return kernelTime + transferTime;
}

#ifdef GRID_SYNC
float srad_gridsync(ResultDatabase &resultDB, OptionParser &op, float* matrix, int imageSize,
          int speckleSize, int iters) {
    kernelTime = 0.0f;
    transferTime = 0.0f;
    int rows, cols, size_I, size_R, niter, iter;
    float *I, *J, lambda, q0sqr, sum, sum2, tmp, meanROI, varROI;


#ifdef GPU

  float *J_cuda;
  float *C_cuda;
  float *E_C, *W_C, *N_C, *S_C;

#endif

  unsigned int r1, r2, c1, c2;
  float *c;

  rows = imageSize;  // number of rows in the domain
  cols = imageSize;  // number of cols in the domain
  if ((rows % 16 != 0) || (cols % 16 != 0)) {
    fprintf(stderr, "rows and cols must be multiples of 16\n");
    exit(1);
  }
  r1 = 0;            // y1 position of the speckle
  r2 = speckleSize;  // y2 position of the speckle
  c1 = 0;            // x1 position of the speckle
  c2 = speckleSize;  // x2 position of the speckle
  lambda = 0.5;      // Lambda value
  niter = iters;     // number of iterations

  size_I = cols * rows;
  size_R = (r2 - r1 + 1) * (c2 - c1 + 1);

  I = (float *)malloc(size_I * sizeof(float));
  J = (float *)malloc(size_I * sizeof(float));
  c = (float *)malloc(sizeof(float) * size_I);

#ifdef GPU

  // Allocate device memory
  CUDA_SAFE_CALL(hipMalloc((void **)&J_cuda, sizeof(float) * size_I));
  CUDA_SAFE_CALL(hipMalloc((void **)&C_cuda, sizeof(float) * size_I));
  CUDA_SAFE_CALL(hipMalloc((void **)&E_C, sizeof(float) * size_I));
  CUDA_SAFE_CALL(hipMalloc((void **)&W_C, sizeof(float) * size_I));
  CUDA_SAFE_CALL(hipMalloc((void **)&S_C, sizeof(float) * size_I));
  CUDA_SAFE_CALL(hipMalloc((void **)&N_C, sizeof(float) * size_I));

#endif

  // Generate a random matrix
  memcpy(I, matrix, rows*cols*sizeof(float));

  for (int k = 0; k < size_I; k++) {
    J[k] = (float)exp(I[k]);
  }
  for (iter = 0; iter < niter; iter++) {
    sum = 0;
    sum2 = 0;
    for (int i = r1; i <= r2; i++) {
      for (int j = c1; j <= c2; j++) {
        tmp = J[i * cols + j];
        sum += tmp;
        sum2 += tmp * tmp;
      }
    }
    meanROI = sum / size_R;
    varROI = (sum2 / size_R) - meanROI * meanROI;
    q0sqr = varROI / (meanROI * meanROI);

#ifdef GPU
    // Currently the input size must be divided by 16 - the block size
    int block_x = cols / BLOCK_SIZE;
    int block_y = rows / BLOCK_SIZE;

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(block_x, block_y);

    // Copy data from main memory to device memory
    hipEventRecord(start, 0);
    CUDA_SAFE_CALL(
        hipMemcpy(J_cuda, J, sizeof(float) * size_I, hipMemcpyHostToDevice));
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    transferTime += elapsed * 1.e-3;

    // Run kernels
    hipEventRecord(start, 0);
    srad_cuda_3<<<dimGrid, dimBlock>>>(E_C, W_C, N_C, S_C, J_cuda, C_cuda, cols,
                                       rows, lambda, q0sqr);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    kernelTime += elapsed * 1.e-3;
    CHECK_CUDA_ERROR();

    // Copy data from device memory to main memory
    hipEventRecord(start, 0);
    CUDA_SAFE_CALL(
        hipMemcpy(J, J_cuda, sizeof(float) * size_I, hipMemcpyDeviceToHost));
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    transferTime += elapsed * 1.e-3;
#endif
  }

    char atts[1024];
    sprintf(atts, "img:%d,speckle:%d,iter:%d", imageSize, speckleSize, iters);
    resultDB.AddResult("srad_gridsync_kernel_time", atts, "sec", kernelTime);
    resultDB.AddResult("srad_gridsync_transer_time", atts, "sec", transferTime);
    resultDB.AddResult("srad_gridsync_total_time", atts, "sec", kernelTime + transferTime);
    resultDB.AddResult("srad_gridsync_parity", atts, "N", transferTime / kernelTime);

  // validate result with result obtained by gridsync
  for (int i = 0; i < rows; i++) {
      for (int j = 0; j < cols; j++) {
          if(check[i*cols+j] - J[i*cols+j] < 0.001) {
              // known bug: with and without gridsync have 10e-5 difference in row 16
              printf("Error: Validation failed at row %d, col %d\n", i, j);
              printf("%0.6f vs %0.6f\n", check[i*cols+j], J[i*cols+j]);
              return FLT_MAX;
          }
      }
  }

  free(I);
  free(J);
  free(c);
#ifdef GPU
  CUDA_SAFE_CALL(hipFree(C_cuda));
  CUDA_SAFE_CALL(hipFree(J_cuda));
  CUDA_SAFE_CALL(hipFree(E_C));
  CUDA_SAFE_CALL(hipFree(W_C));
  CUDA_SAFE_CALL(hipFree(N_C));
  CUDA_SAFE_CALL(hipFree(S_C));
#endif
    return kernelTime + transferTime;
}

#endif //GRID_SYNC

void random_matrix(float *I, int rows, int cols) {
  for (int i = 0; i < rows; i++) {
    for (int j = 0; j < cols; j++) {
      I[i * cols + j] = rand() / (float)RAND_MAX;
    }
  }
}

