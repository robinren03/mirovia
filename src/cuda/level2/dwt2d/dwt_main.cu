/* 
 * Copyright (c) 2009, Jiri Matela
 * All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 * 
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

#include <unistd.h>
#include <error.h>
#include <stdio.h>
#include <stdlib.h>
#include <fcntl.h>
#include <errno.h>
#include <string.h>
#include <assert.h>
#include <sys/time.h>
#include <getopt.h>

#include "ResultDatabase.h"
#include "OptionParser.h"
#include "cudacommon.h"
#include "common.h"
#include "components.h"
#include "dwt.h"
#include "data/create.cpp"

struct dwt {
    char * srcFilename;
    char * outFilename;
    unsigned char *srcImg;
    int pixWidth;
    int pixHeight;
    int components;
    int dwtLvls;
};

int getImg(char * srcFilename, unsigned char *srcImg, int inputSize, bool quiet)
{
    int i = open(srcFilename, O_RDONLY, 0644);
    if (i == -1) { 
        error(0,errno,"Error: cannot access %s", srcFilename);
        return -1;
    }
    int ret = read(i, srcImg, inputSize);
    close(i);

    if(!quiet) {
        printf("precteno %d, inputsize %d\n", ret, inputSize);
    }

    return 0;
}

template <typename T>
void processDWT(struct dwt *d, int forward, int writeVisual, ResultDatabase &resultDB, bool verbose, bool quiet, bool lastPass)
{
    // times
    float transferTime = 0;
    float kernelTime = 0;

    int componentSize = d->pixWidth*d->pixHeight*sizeof(T); T *c_r_out, *backup ;
    CUDA_SAFE_CALL(hipMalloc((void**)&c_r_out, componentSize));
    CUDA_SAFE_CALL(hipMemset(c_r_out, 0, componentSize));
    
    CUDA_SAFE_CALL(hipMalloc((void**)&backup, componentSize));
    CUDA_SAFE_CALL(hipMemset(backup, 0, componentSize));
	
    if (d->components == 3) {
        /* Alloc two more buffers for G and B */
        T *c_g_out, *c_b_out;
        CUDA_SAFE_CALL(hipMalloc((void**)&c_g_out, componentSize));
        CUDA_SAFE_CALL(hipMemset(c_g_out, 0, componentSize));
        
        CUDA_SAFE_CALL(hipMalloc((void**)&c_b_out, componentSize));
        CUDA_SAFE_CALL(hipMemset(c_b_out, 0, componentSize));
        
        /* Load components */
        T *c_r, *c_g, *c_b;
        // R, aligned component size
        CUDA_SAFE_CALL(hipMalloc((void**)&c_r, componentSize)); 
        CUDA_SAFE_CALL(hipMemset(c_r, 0, componentSize));
        // G, aligned component size
        CUDA_SAFE_CALL(hipMalloc((void**)&c_g, componentSize)); 
        CUDA_SAFE_CALL(hipMemset(c_g, 0, componentSize));
        // B, aligned component size
        CUDA_SAFE_CALL(hipMalloc((void**)&c_b, componentSize));
        CUDA_SAFE_CALL(hipMemset(c_b, 0, componentSize));

        rgbToComponents(c_r, c_g, c_b, d->srcImg, d->pixWidth, d->pixHeight, transferTime, kernelTime);
        /* Compute DWT and always store into file */
        nStage2dDWT(c_r, c_r_out, backup, d->pixWidth, d->pixHeight, d->dwtLvls, forward, transferTime, kernelTime, verbose, quiet);
        nStage2dDWT(c_g, c_g_out, backup, d->pixWidth, d->pixHeight, d->dwtLvls, forward, transferTime, kernelTime, verbose, quiet);
        nStage2dDWT(c_b, c_b_out, backup, d->pixWidth, d->pixHeight, d->dwtLvls, forward, transferTime, kernelTime, verbose, quiet);

        // -------test----------
        // T *h_r_out=(T*)malloc(componentSize);
		// hipMemcpy(h_r_out, c_g_out, componentSize, hipMemcpyDeviceToHost);
        // int ii;
		// for(ii=0;ii<componentSize/sizeof(T);ii++) {
			// fprintf(stderr, "%d ", h_r_out[ii]);
			// if((ii+1) % (d->pixWidth) == 0) fprintf(stderr, "\n");
        // }
        // -------test----------
        
		
        /* Store DWT to file */
        if (writeVisual) {
            writeNStage2DDWT(c_r_out, d->pixWidth, d->pixHeight, d->dwtLvls, d->outFilename, ".r");
            writeNStage2DDWT(c_g_out, d->pixWidth, d->pixHeight, d->dwtLvls, d->outFilename, ".g");
            writeNStage2DDWT(c_b_out, d->pixWidth, d->pixHeight, d->dwtLvls, d->outFilename, ".b");
        } else {
            writeLinear(c_r_out, d->pixWidth, d->pixHeight, d->outFilename, ".r");
            writeLinear(c_g_out, d->pixWidth, d->pixHeight, d->outFilename, ".g");
            writeLinear(c_b_out, d->pixWidth, d->pixHeight, d->outFilename, ".b");
        }
        if(lastPass && !quiet) {
            printf("Writing to %s.r (%d x %d)\n", d->outFilename, d->pixWidth, d->pixHeight);
            printf("Writing to %s.g (%d x %d)\n", d->outFilename, d->pixWidth, d->pixHeight);
            printf("Writing to %s.b (%d x %d)\n", d->outFilename, d->pixWidth, d->pixHeight);
        }
            

        hipFree(c_r);
        hipFree(c_g);
        hipFree(c_b);
        hipFree(c_g_out);
        hipFree(c_b_out);

    } 
    else if (d->components == 1) {
        //Load component
        T *c_r;
        // R, aligned component size
        CUDA_SAFE_CALL(hipMalloc((void**)&(c_r), componentSize)); 
        CUDA_SAFE_CALL(hipMemset(c_r, 0, componentSize));

        bwToComponent(c_r, d->srcImg, d->pixWidth, d->pixHeight, transferTime, kernelTime);

        // Compute DWT 
        nStage2dDWT(c_r, c_r_out, backup, d->pixWidth, d->pixHeight, d->dwtLvls, forward, transferTime, kernelTime, verbose, quiet);

        // Store DWT to file 
        if (writeVisual) {
            writeNStage2DDWT(c_r_out, d->pixWidth, d->pixHeight, d->dwtLvls, d->outFilename, ".out");
            if(lastPass && !quiet) {
                printf("Writing to %s.out (%d x %d)\n", d->outFilename, d->pixWidth, d->pixHeight);
            }
        } else {
            writeLinear(c_r_out, d->pixWidth, d->pixHeight, d->outFilename, ".lin.out");
            if(lastPass && !quiet) {
                printf("Writing to %s.lin.out (%d x %d)\n", d->outFilename, d->pixWidth, d->pixHeight);
            }
        }
        hipFree(c_r);
    }

    hipFree(c_r_out);
    hipFree(backup);

    char atts[16];
    sprintf(atts, "%dx%d", d->pixWidth, d->pixHeight);
    resultDB.AddResult("dwt_kernel_time", atts, "sec", kernelTime);
    resultDB.AddResult("dwt_transfer_time", atts, "sec", transferTime);
    resultDB.AddResult("dwt_total_time", atts, "sec", kernelTime+transferTime);
    resultDB.AddResult("dwt_parity", atts, "N", transferTime/kernelTime);
    resultDB.AddOverall("Time", "sec", kernelTime+transferTime);
}

void addBenchmarkSpecOptions(OptionParser &op) {
    op.addOption("pixWidth", OPT_INT, "1", "real pixel width");
    op.addOption("pixHeight", OPT_INT, "1", "real pixel height");
    op.addOption("compCount", OPT_INT, "3", "number of components (3 for RGB/YUV, 4 for RGBA");
    op.addOption("bitDepth", OPT_INT, "8", "bit depth of src img");
    op.addOption("levels", OPT_INT, "3", "number of DWT levels");
    op.addOption("reverse", OPT_BOOL, "0", "reverse transform (defaults to forward");
    op.addOption("53", OPT_BOOL, "0", "5/3 transform (defaults to 9/7)");
    op.addOption("writeVisual", OPT_BOOL, "0", "write output in visual (tiled) order instead of linear");
}

void RunBenchmark(ResultDatabase &resultDB, OptionParser &op)
{
    printf("Running DWT2D\n");
    bool quiet      = op.getOptionBool("quiet");
    bool verbose    = op.getOptionBool("verbose");
    int pixWidth    = op.getOptionInt("pixWidth"); //<real pixWidth
    int pixHeight   = op.getOptionInt("pixHeight"); //<real pixHeight
    int compCount   = op.getOptionInt("compCount"); //number of components; 3 for RGB or YUV, 4 for RGBA
    int bitDepth    = op.getOptionInt("bitDepth");; 
    int dwtLvls     = op.getOptionInt("levels"); //default numuber of DWT levels
    bool forward     = !op.getOptionBool("reverse"); //forward transform
    bool dwt97       = !op.getOptionBool("53"); //1=dwt9/7, 0=dwt5/3 transform
    bool writeVisual = op.getOptionBool("writeVisual"); //write output (subbands) in visual (tiled) order instead of linear
    string inputFile = op.getOptionString("inputFile");
    if(inputFile.empty()) {
        int probSizes[4] = {48, 192, 8192, 2<<13};
        int pix = probSizes[op.getOptionInt("size")-1];
        inputFile = datagen(pix);
        pixWidth = pix;
        pixHeight = pix;
    }

    if (pixWidth <= 0 || pixHeight <=0) {
        printf("Wrong or missing dimensions\n");
        return;
    }

    if (forward == 0) {
        writeVisual = 0; //do not write visual when RDWT
    }

    struct dwt *d;
    d = (struct dwt *)malloc(sizeof(struct dwt));
    d->srcImg = NULL;
    d->pixWidth = pixWidth;
    d->pixHeight = pixHeight;
    d->components = compCount;
    d->dwtLvls  = dwtLvls;

    // file names
    d->srcFilename = (char*)malloc(strlen(inputFile.c_str()));
    strcpy(d->srcFilename, inputFile.c_str());
    d->outFilename = (char *)malloc(strlen(d->srcFilename)+4);
    strcpy(d->outFilename, d->srcFilename);
    strcpy(d->outFilename+strlen(d->srcFilename), ".dwt");

    //Input review
    if(!quiet) {
        printf("Source file:\t\t%s\n", d->srcFilename);
        printf(" Dimensions:\t\t%dx%d\n", pixWidth, pixHeight);
        printf(" Components count:\t%d\n", compCount);
        printf(" Bit depth:\t\t%d\n", bitDepth);
        printf(" DWT levels:\t\t%d\n", dwtLvls);
        printf(" Forward transform:\t%d\n", forward);
        printf(" 9/7 transform:\t\t%d\n", dwt97);
        printf(" Write visual:\t\t%d\n", writeVisual);
    }
    
    //data sizes
    int inputSize = pixWidth*pixHeight*compCount; //<amount of data (in bytes) to proccess

    //load img source image
    CUDA_SAFE_CALL(hipHostMalloc((void **)&d->srcImg, inputSize));
    if (getImg(d->srcFilename, d->srcImg, inputSize, quiet) == -1) 
        return;

    int passes = op.getOptionInt("passes");
    for(int i = 0; i < passes; i++) {
        bool lastPass = i+1 == passes;
        if(!quiet) {
            printf("Pass %d:\n", i);
        }
        /* DWT */
        if (forward == 1) {
            if(dwt97 == 1 ) {
                processDWT<float>(d, forward, writeVisual, resultDB, verbose, quiet, lastPass);
            } else { // 5/3
                processDWT<int>(d, forward, writeVisual, resultDB, verbose, quiet, lastPass);
            }
        }
        else { // reverse
            if(dwt97 == 1 ) {
                processDWT<float>(d, forward, writeVisual, resultDB, verbose, quiet, lastPass);
            } else { // 5/3
                processDWT<int>(d, forward, writeVisual, resultDB, verbose, quiet, lastPass);
            }
        }
        if(!quiet) {
            printf("Done.\n");
        }
    }

    //writeComponent(r_cuda, pixWidth, pixHeight, srcFilename, ".g");
    //writeComponent(g_wave_cuda, 512000, ".g");
    //writeComponent(g_cuda, componentSize, ".g");
    //writeComponent(b_wave_cuda, componentSize, ".b");
    CUDA_SAFE_CALL(hipHostFree(d->srcImg));
}
