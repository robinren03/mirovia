#include "hip/hip_runtime.h"
/***********************************************************************************
  Implementing Breadth first search on CUDA using algorithm given in HiPC'07
  paper "Accelerating Large Graph Algorithms on the GPU using CUDA"

  Copyright (c) 2008 International Institute of Information Technology - Hyderabad. 
  All rights reserved.

  Permission to use, copy, modify and distribute this software and its documentation for 
  educational purpose is hereby granted without fee, provided that the above copyright 
  notice and this permission notice appear in all copies of this software and that you do 
  not sell the software.

  THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR 
  OTHERWISE.

  Created by Pawan Harish.
 ************************************************************************************/
#include <stdlib.h>
#include <stdio.h>
#include <string>
#include <cassert>
#include <math.h>
#include <hip/hip_runtime.h>

#include "cudacommon.h"
#include "ResultDatabase.h"
#include "OptionParser.h"

#define MIN_NODES 20
#define MAX_NODES ULONG_MAX
#define MIN_EDGES 2
#define MAX_INIT_EDGES 4 // Nodes will have, on average, 2*MAX_INIT_EDGES edges
#define MIN_WEIGHT 1
#define MAX_WEIGHT 10
#define SEED 7

#define MAX_THREADS_PER_BLOCK 256

using namespace std;

struct Node
{
	int starting;
	int no_of_edges;
};

void initGraph(OptionParser &op, int &no_of_nodes, int &edge_list_size, int &source, Node* &h_graph_nodes, int* &h_graph_edges);
float BFSGraph(ResultDatabase &resultDB, OptionParser &op, int no_of_nodes, int edge_list_size, int source, Node* &h_graph_nodes, int* &h_graph_edges);
#ifdef UNIFIED_MEMORY
float BFSGraphUnifiedMemory(ResultDatabase &resultDB, OptionParser &op, int no_of_nodes, int edge_list_size, int source, Node* &h_graph_nodes, int* &h_graph_edges);
#endif

////////////////////////////////////////////////////////////////////////////////
__global__ void Kernel( Node* g_graph_nodes, int* g_graph_edges, bool* g_graph_mask, bool* g_updating_graph_mask, bool *g_graph_visited, int* g_cost, int no_of_nodes) 
{
	int tid = (blockIdx.x*MAX_THREADS_PER_BLOCK) + threadIdx.x;
	if( tid<no_of_nodes && g_graph_mask[tid])
	{
		g_graph_mask[tid]=false;
		for(int i=g_graph_nodes[tid].starting; i<(g_graph_nodes[tid].no_of_edges + g_graph_nodes[tid].starting); i++)
			{
			int id = g_graph_edges[i];
			if(!g_graph_visited[id])
				{
				g_cost[id]=g_cost[tid]+1;
				g_updating_graph_mask[id]=true;
				}
			}
	}
}

__global__ void Kernel2( bool* g_graph_mask, bool *g_updating_graph_mask, bool* g_graph_visited, bool *g_over, int no_of_nodes)
{
	int tid = (blockIdx.x*MAX_THREADS_PER_BLOCK) + threadIdx.x;
	if( tid<no_of_nodes && g_updating_graph_mask[tid])
	{

		g_graph_mask[tid]=true;
		g_graph_visited[tid]=true;
		*g_over=true;
		g_updating_graph_mask[tid]=false;
	}
}
////////////////////////////////////////////////////////////////////////////////

// ****************************************************************************
// Function: addBenchmarkSpecOptions
//
// Purpose:
//   Add benchmark specific options parsing
//
// Arguments:
//   op: the options parser / parameter database
//
// Returns:  nothing
//
// Programmer: Kyle Spafford
// Creation: August 13, 2009
//
// Modifications:
//
// ****************************************************************************
void addBenchmarkSpecOptions(OptionParser &op) {
}

// ****************************************************************************
// Function: RunBenchmark
//
// Purpose:
//   Executes the radix sort benchmark
//
// Arguments:
//   resultDB: results from the benchmark are stored in this db
//   op: the options parser / parameter database
//
// Returns:  nothing, results are stored in resultDB
//
// Programmer: Kyle Spafford
// Creation: August 13, 2009
//
// Modifications:
//
// ****************************************************************************
void RunBenchmark(ResultDatabase &resultDB, OptionParser &op) {
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);

    // seed random number generator
	srand(SEED);

    int no_of_nodes = 0;
    int edge_list_size = 0;
    int source = 0;
	Node* h_graph_nodes;
	int* h_graph_edges;
    initGraph(op, no_of_nodes, edge_list_size, source, h_graph_nodes, h_graph_edges);

    // atts string for result database
    char tmp[64];
    sprintf(tmp, "%dV,%dE", no_of_nodes, edge_list_size);
    string atts = string(tmp);

    bool quiet = op.getOptionBool("quiet");
    int passes = op.getOptionInt("passes");
    for(int i = 0; i < passes; i++) {
        if(!quiet) {
            printf("Pass %d:\n", i);
        }
        float time = BFSGraph(resultDB, op, no_of_nodes, edge_list_size, source, h_graph_nodes, h_graph_edges);
        if(!quiet) {
            if(time == FLT_MAX) {
                printf("Executing BFS...Error.\n");
            } else {
                printf("Executing BFS...Done.\n");
            }
        }
#ifdef UNIFIED_MEMORY
        float timeUM = BFSGraphUnifiedMemory(resultDB, op, no_of_nodes, edge_list_size, source, h_graph_nodes, h_graph_edges);
        if(!quiet) {
            if(timeUM == FLT_MAX) {
                printf("Executing BFS using unified memory...Error.\n");
            } else {
                printf("Executing BFS using unified memory...Done.\n");
            }
        }
        if(time != FLT_MAX && timeUM != FLT_MAX) {
            resultDB.AddResult("BFS_Time/BFS_UM_Time", atts, "N", time/timeUM);
        }
#endif
    }

	free( h_graph_nodes);
	free( h_graph_edges);
}

////////////////////////////////////////////////////////////////////////////////
//Generate uniform distribution
////////////////////////////////////////////////////////////////////////////////
int uniform_distribution(int rangeLow, int rangeHigh) {
    double myRand = rand()/(1.0 + RAND_MAX); 
    int range = rangeHigh - rangeLow + 1;
    int myRand_scaled = (myRand * range) + rangeLow;
    return myRand_scaled;
}

////////////////////////////////////////////////////////////////////////////////
//Initialize Graph
////////////////////////////////////////////////////////////////////////////////
void initGraph(OptionParser &op, int &no_of_nodes, int &edge_list_size, int &source, Node* &h_graph_nodes, int* &h_graph_edges) {
    bool quiet = op.getOptionBool("quiet");
    // open input file for reading
    FILE *fp = NULL;
    string infile = op.getOptionString("inputFile");
    if(infile != "") {
        fp = fopen(infile.c_str(),"r");
        if(!fp && !quiet)
        {
            printf("Error: Unable to read graph file %s.\n", infile.c_str());
        }
    }

    if(!quiet) {
        if(fp) {
            printf("Reading graph file\n");
        } else {
            printf("Generating graph with problem size %d\n", (int)op.getOptionInt("size"));
        }
    }

    // initialize number of nodes
    if(fp) {
	    int n = fscanf(fp,"%d",&no_of_nodes);
        assert(n == 1);
    } else {
        int problemSizes[4] = {10, 50, 100, 200};
        no_of_nodes = problemSizes[op.getOptionInt("size") - 1] * 1024 * 1024;
    }

	// initalize the nodes & number of edges
	h_graph_nodes = (Node*) malloc(sizeof(Node)*no_of_nodes);
	int start;
    int edgeno;   
    for( int i = 0; i < no_of_nodes; i++) 
    {
        if(fp) {
            int n = fscanf(fp,"%d %d",&start,&edgeno);
            assert(n == 2);
        } else {
            start = edge_list_size;
            edgeno = rand() % (MAX_INIT_EDGES - MIN_EDGES + 1) + MIN_EDGES;
        }
        h_graph_nodes[i].starting = start;
        h_graph_nodes[i].no_of_edges = edgeno;
        edge_list_size += edgeno;
    }

	// initialize the source node
    if(fp) {
	    int n = fscanf(fp,"%d",&source);
        assert(n == 1);
    } else {
        source = uniform_distribution(0, no_of_nodes - 1);
    }
    source = 0;

    if(fp) {
        int edges;
        int n = fscanf(fp,"%d",&edges);
        assert(n == 1);
        assert(edges == edge_list_size);
    }

    // initialize the edges
	int id;
    int cost;
	h_graph_edges = (int*) malloc(sizeof(int)*edge_list_size);
	for(int i=0; i < edge_list_size ; i++)
	{
        if(fp) {
            int n = fscanf(fp,"%d %d",&id, &cost);
            assert(n == 2);
        } else {
			id = uniform_distribution(0, no_of_nodes - 1);
			//cost = rand() % (MAX_WEIGHT - MIN_WEIGHT + 1) + MIN_WEIGHT;
        }
		h_graph_edges[i] = id;
	}

    if(!quiet) {
        if(fp) {
            fclose(fp);    
            printf("Done reading graph file\n");
        } else {
            printf("Done generating graph\n");
        }
        printf("Graph size: %d nodes, %d edges\n", no_of_nodes, edge_list_size);
    }
}

////////////////////////////////////////////////////////////////////////////////
//Apply BFS on a Graph using CUDA
////////////////////////////////////////////////////////////////////////////////
float BFSGraph(ResultDatabase &resultDB, OptionParser &op, int no_of_nodes, int edge_list_size, int source, Node* &h_graph_nodes, int* &h_graph_edges) 
{
    bool verbose = op.getOptionBool("verbose");

	int num_of_blocks = 1;
	int num_of_threads_per_block = no_of_nodes;
	//Make execution Parameters according to the number of nodes
	//Distribute threads across multiple Blocks if necessary
	if(no_of_nodes>MAX_THREADS_PER_BLOCK)
	{
		num_of_blocks = (int)ceil(no_of_nodes/(double)MAX_THREADS_PER_BLOCK);
		num_of_threads_per_block = MAX_THREADS_PER_BLOCK; 
	}

	// allocate host memory
	bool *h_graph_mask = (bool*) malloc(sizeof(bool)*no_of_nodes);
	bool *h_updating_graph_mask = (bool*) malloc(sizeof(bool)*no_of_nodes);
	bool *h_graph_visited = (bool*) malloc(sizeof(bool)*no_of_nodes);

	// initalize the memory
    for( int i = 0; i < no_of_nodes; i++) 
    {
        h_graph_mask[i]=false;
        h_updating_graph_mask[i]=false;
        h_graph_visited[i]=false;
    }

	//set the source node as true in the mask
	h_graph_mask[source]=true;
	h_graph_visited[source]=true;

	// allocate mem for the result on host side
	int* h_cost = (int*) malloc( sizeof(int)*no_of_nodes);
	for(int i=0;i<no_of_nodes;i++) {
		h_cost[i]=-1;
    }
	h_cost[source]=0;

	// node list
	Node* d_graph_nodes;
	// edge list
	int* d_graph_edges;
	// mask
	bool* d_graph_mask;
	bool* d_updating_graph_mask;
	// visited nodes
	bool* d_graph_visited;
    // result
	int* d_cost;
	// bool if execution is over
	bool *d_over;

	CUDA_SAFE_CALL_NOEXIT(hipMalloc( (void**) &d_graph_nodes, sizeof(Node)*no_of_nodes));
	CUDA_SAFE_CALL_NOEXIT(hipMalloc( (void**) &d_graph_edges, sizeof(int)*edge_list_size));
	CUDA_SAFE_CALL_NOEXIT(hipMalloc( (void**) &d_graph_mask, sizeof(bool)*no_of_nodes));
	CUDA_SAFE_CALL_NOEXIT(hipMalloc( (void**) &d_updating_graph_mask, sizeof(bool)*no_of_nodes));
	CUDA_SAFE_CALL_NOEXIT(hipMalloc( (void**) &d_graph_visited, sizeof(bool)*no_of_nodes));
	CUDA_SAFE_CALL_NOEXIT(hipMalloc( (void**) &d_cost, sizeof(int)*no_of_nodes));
	CUDA_SAFE_CALL_NOEXIT(hipMalloc( (void**) &d_over, sizeof(bool)));
    hipError_t err = hipGetLastError();
    if(err != hipSuccess) {
        free( h_graph_mask);
        free( h_updating_graph_mask);
        free( h_graph_visited);
        free( h_cost);
        hipFree(d_graph_nodes);
        hipFree(d_graph_edges);
        hipFree(d_graph_mask);
        hipFree(d_updating_graph_mask);
        hipFree(d_graph_visited);
        hipFree(d_cost);  
        hipFree(d_over);
        return FLT_MAX;
    }

    hipEvent_t tstart, tstop;
    hipEventCreate(&tstart);
    hipEventCreate(&tstop);
    float elapsedTime;
double transferTime = 0.;
    hipEventRecord(tstart, 0);
	hipMemcpy( d_graph_nodes, h_graph_nodes, sizeof(Node)*no_of_nodes, hipMemcpyHostToDevice) ;
	hipMemcpy( d_graph_edges, h_graph_edges, sizeof(int)*edge_list_size, hipMemcpyHostToDevice) ;
	hipMemcpy( d_graph_mask, h_graph_mask, sizeof(bool)*no_of_nodes, hipMemcpyHostToDevice) ;
	hipMemcpy( d_updating_graph_mask, h_updating_graph_mask, sizeof(bool)*no_of_nodes, hipMemcpyHostToDevice) ;
	hipMemcpy( d_graph_visited, h_graph_visited, sizeof(bool)*no_of_nodes, hipMemcpyHostToDevice) ;
	hipMemcpy( d_cost, h_cost, sizeof(int)*no_of_nodes, hipMemcpyHostToDevice) ;
    hipEventRecord(tstop, 0);
    hipEventSynchronize(tstop);
    hipEventElapsedTime(&elapsedTime, tstart, tstop);
    transferTime += elapsedTime * 1.e-3; // convert to seconds

	// setup execution parameters
	dim3  grid( num_of_blocks, 1, 1);
	dim3  threads( num_of_threads_per_block, 1, 1);

    double kernelTime = 0;
	int k=0;
	bool stop;
	//Call the Kernel untill all the elements of Frontier are not false
	do
	{
		//if no thread changes this value then the loop stops
		stop=false;
		hipMemcpy( d_over, &stop, sizeof(bool), hipMemcpyHostToDevice) ;

        hipEventRecord(tstart, 0);
        Kernel<<< grid, threads, 0 >>>( d_graph_nodes, d_graph_edges, d_graph_mask, d_updating_graph_mask, d_graph_visited, d_cost, no_of_nodes);
        hipEventRecord(tstop, 0);
        hipEventSynchronize(tstop);
        hipEventElapsedTime(&elapsedTime, tstart, tstop);
        kernelTime += elapsedTime * 1.e-3;
        CHECK_CUDA_ERROR();

        // check if kernel execution generated an error
        hipEventRecord(tstart, 0);
        Kernel2<<< grid, threads, 0 >>>( d_graph_mask, d_updating_graph_mask, d_graph_visited, d_over, no_of_nodes);
        hipEventRecord(tstop, 0);
        hipEventSynchronize(tstop);
        hipEventElapsedTime(&elapsedTime, tstart, tstop);
        kernelTime += elapsedTime * 1.e-3;
        CHECK_CUDA_ERROR()

        hipMemcpy( &stop, d_over, sizeof(bool), hipMemcpyDeviceToHost) ;

		k++;
	}
	while(stop);

    if(verbose) {
	    printf("Kernel Executed %d times\n",k);
    }

	// copy result from device to host
    hipEventRecord(tstart, 0);
	hipMemcpy( h_cost, d_cost, sizeof(int)*no_of_nodes, hipMemcpyDeviceToHost) ;
    hipEventRecord(tstop, 0);
    hipEventSynchronize(tstop);
    hipEventElapsedTime(&elapsedTime, tstart, tstop);
    transferTime += elapsedTime * 1.e-3; // convert to seconds

	//Store the result into a file
    string outfile = op.getOptionString("outputFile");
    if(outfile != "") {
        FILE *fpo = fopen(outfile.c_str(),"w");
        for(int i=0;i<no_of_nodes;i++) {
            fprintf(fpo,"%d) cost:%d\n",i,h_cost[i]);
        }
        fclose(fpo);
    }

	// cleanup memory
	free( h_graph_mask);
	free( h_updating_graph_mask);
	free( h_graph_visited);
	free( h_cost);
	hipFree(d_graph_nodes);
	hipFree(d_graph_edges);
	hipFree(d_graph_mask);
	hipFree(d_updating_graph_mask);
	hipFree(d_graph_visited);
	hipFree(d_cost);  
    hipFree(d_over);

    char tmp[64];
    sprintf(tmp, "%dV,%dE", no_of_nodes, edge_list_size);
    string atts = string(tmp);
    resultDB.AddResult("BFS-TransferTime", atts, "sec", transferTime);
    resultDB.AddResult("BFS-KernelTime", atts, "sec", kernelTime);
    resultDB.AddResult("BFS-TotalTime", atts, "sec", transferTime + kernelTime);
    resultDB.AddResult("BFS-Rate_Nodes", atts, "Nodes/s", no_of_nodes/kernelTime);
    resultDB.AddResult("BFS-Rate_Edges", atts, "Edges/s", edge_list_size/kernelTime);
    resultDB.AddResult("BFS-Rate_Parity", atts, "N", transferTime / kernelTime);
    return transferTime + kernelTime;
}

#ifdef UNIFIED_MEMORY
////////////////////////////////////////////////////////////////////////////////
//Apply BFS on a Graph using CUDA and Unified Memory
////////////////////////////////////////////////////////////////////////////////
float BFSGraphUnifiedMemory(ResultDatabase &resultDB, OptionParser &op, int no_of_nodes, int edge_list_size, int source, Node* &h_graph_nodes, int* &h_graph_edges) {
    bool verbose = op.getOptionBool("verbose");
    bool quiet = op.getOptionBool("quiet");

	int num_of_blocks = 1;
	int num_of_threads_per_block = no_of_nodes;
	//Make execution Parameters according to the number of nodes
	//Distribute threads across multiple Blocks if necessary
	if(no_of_nodes>MAX_THREADS_PER_BLOCK)
	{
		num_of_blocks = (int)ceil(no_of_nodes/(double)MAX_THREADS_PER_BLOCK);
		num_of_threads_per_block = MAX_THREADS_PER_BLOCK; 
	}

    // copy graph nodes to unified memory
    Node* graph_nodes;
    CUDA_SAFE_CALL(hipMallocManaged(&graph_nodes, sizeof(Node)*no_of_nodes));
    memcpy(graph_nodes, h_graph_nodes, sizeof(Node)*no_of_nodes);

    // copy graph edges to unified memory
    int* graph_edges;
    CUDA_SAFE_CALL(hipMallocManaged(&graph_edges, sizeof(int)*edge_list_size));
    memcpy(graph_edges, h_graph_edges, sizeof(int)*edge_list_size);

	// allocate and initalize the memory
    bool* graph_mask;
    bool* updating_graph_mask;
    bool* graph_visited;
    CUDA_SAFE_CALL(hipMallocManaged(&graph_mask, sizeof(bool)*no_of_nodes));
    CUDA_SAFE_CALL(hipMallocManaged(&updating_graph_mask, sizeof(bool)*no_of_nodes));
    CUDA_SAFE_CALL(hipMallocManaged(&graph_visited, sizeof(bool)*no_of_nodes));
    hipError_t err = hipGetLastError();
    for( int i = 0; i < no_of_nodes; i++) 
    {
        graph_mask[i]=false;
        updating_graph_mask[i]=false;
        graph_visited[i]=false;
    }

	//set the source node as true in the mask
	graph_mask[source]=true;
	graph_visited[source]=true;

    // allocate and initialize memory for result
    int* cost;
    CUDA_SAFE_CALL(hipMallocManaged(&cost, sizeof(int)*no_of_nodes));
    if(err != hipSuccess) {
        hipFree(graph_nodes);
        hipFree(graph_edges);
        hipFree(graph_mask);
        hipFree(updating_graph_mask);
        hipFree(graph_visited);
        hipFree(cost);
        return FLT_MAX;
    }

	for(int i=0;i<no_of_nodes;i++) {
		cost[i]=-1;
    }
	cost[source]=0;

	// bool if execution is over
    bool* over;
    CUDA_SAFE_CALL(hipMallocManaged(&over, sizeof(bool)));

    // events for timing
    hipEvent_t tstart, tstop;
    hipEventCreate(&tstart);
    hipEventCreate(&tstop);
    float elapsedTime;

	// setup execution parameters
	dim3  grid( num_of_blocks, 1, 1);
	dim3  threads( num_of_threads_per_block, 1, 1);

    double kernelTime = 0;
	int k=0;
    bool stop;
	//Call the Kernel until all the elements of Frontier are not false
	do
	{
        stop = false;
        *over = stop;

        hipEventRecord(tstart, 0);
        Kernel<<< grid, threads, 0 >>>(graph_nodes, graph_edges, graph_mask, updating_graph_mask, graph_visited, cost, no_of_nodes);
        hipEventRecord(tstop, 0);
        hipEventSynchronize(tstop);
        hipEventElapsedTime(&elapsedTime, tstart, tstop);
        kernelTime += elapsedTime * 1.e-3;
        CHECK_CUDA_ERROR();

        // check if kernel execution generated an error
        hipEventRecord(tstart, 0);
        Kernel2<<< grid, threads, 0 >>>(graph_mask, updating_graph_mask, graph_visited, over, no_of_nodes);
        hipEventRecord(tstop, 0);
        hipEventSynchronize(tstop);
        hipEventElapsedTime(&elapsedTime, tstart, tstop);
        kernelTime += elapsedTime * 1.e-3;
        CHECK_CUDA_ERROR()

        stop = *over;
		k++;
	}
	while(stop);

    if(verbose && !quiet) {
        printf("Kernel Time: %f\n", kernelTime);
        printf("Kernel Executed %d times\n",k);
    }

    // cleanup memory
	hipFree(graph_nodes);
	hipFree(graph_edges);
	hipFree(graph_mask);
	hipFree(updating_graph_mask);
	hipFree(graph_visited);
	hipFree(cost);
    hipFree(over);

    char tmp[64];
    sprintf(tmp, "%dV,%dE", no_of_nodes, edge_list_size);
    string atts = string(tmp);
    resultDB.AddResult("BFS-UM-TotalTime", atts, "sec", kernelTime);
    resultDB.AddResult("BFS-UM-Rate_Nodes", atts, "Nodes/s", no_of_nodes/kernelTime);
    resultDB.AddResult("BFS-UM-Rate_Edges", atts, "Edges/s", edge_list_size/kernelTime);
    return kernelTime;
}
#endif
