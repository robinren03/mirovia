#include "hip/hip_runtime.h"
void BFSGraphManagedMemory(ResultDatabase &resultDB, OptionParser &op, int no_of_nodes, int edge_list_size, int source, Node* &h_graph_nodes, int* &h_graph_edges) {
    bool verbose = op.getOptionBool("verbose");
	int num_of_blocks = 1;
	int num_of_threads_per_block = no_of_nodes;
	//Make execution Parameters according to the number of nodes
	//Distribute threads across multiple Blocks if necessary
	if(no_of_nodes>MAX_THREADS_PER_BLOCK)
	{
		num_of_blocks = (int)ceil(no_of_nodes/(double)MAX_THREADS_PER_BLOCK); 
		num_of_threads_per_block = MAX_THREADS_PER_BLOCK; 
	}

    // copy graph nodes to unified memory
    Node* graph_nodes;
    CUDA_SAFE_CALL(hipMallocManaged(&graph_nodes, sizeof(Node)*no_of_nodes));
    memcpy(graph_nodes, h_graph_nodes, sizeof(Node)*no_of_nodes);
    // copy graph edges to unified memory
    int* graph_edges;
    CUDA_SAFE_CALL(hipMallocManaged(&graph_edges, sizeof(int)*edge_list_size));
    memcpy(graph_edges, h_graph_edges, sizeof(int)*edge_list_size);

	// allocate and initalize the memory
    bool* graph_mask, updating_graph_mask, graph_visited;
    CUDA_SAFE_CALL(hipMallocManaged(&graph_mask, sizeof(bool)*no_of_nodes));
    CUDA_SAFE_CALL(hipMallocManaged(&updating_graph_mask, sizeof(bool)*no_of_nodes));
    CUDA_SAFE_CALL(hipMallocManaged(&graph_visited, sizeof(bool)*no_of_nodes));
    for( unsigned int i = 0; i < no_of_nodes; i++) 
    {
        graph_mask[i]=false;
        updating_graph_mask[i]=false;
        graph_visited[i]=false;
    }

	//set the source node as true in the mask
	graph_mask[source]=true;
	graph_visited[source]=true;

    // allocate and initialize memory for result
    int* cost;
    CUDA_SAFE_CALL(hipMallocManaged(&cost, sizeof(int)*no_of_nodes);
	for(int i=0;i<no_of_nodes;i++) {
		cost[i]=-1;
    }
	cost[source]=0;

	// bool if execution is over
    bool* over;
    CUDA_SAFE_CALL(hipMallocManaged(&over, sizeof(bool));

    // events for timing
    hipEvent_t tstart, tstop;
    hipEventCreate(&tstart);
    hipEventCreate(&tstop);
    float elapsedTime;

	// setup execution parameters
	dim3  grid( num_of_blocks, 1, 1);
	dim3  threads( num_of_threads_per_block, 1, 1);

    double kernelTime = 0;
	int k=0;
	bool stop;
	//Call the Kernel untill all the elements of Frontier are not false
	do
	{
		//if no thread changes this value then the loop stops
		stop=false;
        stop = &over;

        hipEventRecord(tstart, 0);
        Kernel<<< grid, threads, 0 >>>(graph_nodes, graph_edges, graph_mask, updating_graph_mask, graph_visited, cost, no_of_nodes);
        hipEventRecord(tstop, 0);
        hipEventSynchronize(tstop);
        hipEventElapsedTime(&elapsedTime, tstart, tstop);
        kernelTime += elapsedTime * 1.e-3;
        CHECK_CUDA_ERROR();

        // check if kernel execution generated an error
        hipEventRecord(tstart, 0);
        Kernel2<<< grid, threads, 0 >>>(graph_mask, updating_graph_mask, graph_visited, over, no_of_nodes);
        hipEventRecord(tstop, 0);
        hipEventSynchronize(tstop);
        hipEventElapsedTime(&elapsedTime, tstart, tstop);
        kernelTime += elapsedTime * 1.e-3;
        CHECK_CUDA_ERROR()

		k++;
	}
	while(stop);

    printf("Kernel Time: %f\n", kernelTime);
    printf("Kernel Executed %d times\n",k);

	hipFree(graph_nodes);
	hipFree(graph_edges);
	hipFree(graph_mask);
	hipFree(updating_graph_mask);
	hipFree(graph_visited);
	hipFree(cost);
    hipFree(over);
}

