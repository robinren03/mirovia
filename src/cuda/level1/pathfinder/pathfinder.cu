#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <fstream>

#include "OptionParser.h"
#include "ResultDatabase.h"
#include "Utility.h"
#include "cudacommon.h"

#define BLOCK_SIZE 512
#define STR_SIZE 256
#define HALO 1  // halo width along one direction when advancing to the next iteration
#define SEED 7

void run(int borderCols, int smallBlockCol, int blockCols, ResultDatabase &resultDB, OptionParser &op);

int rows, cols;
int *data;
int **wall;
int *result;
int pyramid_height;

// ****************************************************************************
// Function: addBenchmarkSpecOptions
//
// Purpose:
//   Add benchmark specific options parsing.  The user is allowed to specify
//   the size of the input data in kiB.
//
// Arguments:
//   op: the options parser / parameter database
//
// Programmer: Anthony Danalis
// Creation: September 08, 2009
// Returns:  nothing
//
// ****************************************************************************
void addBenchmarkSpecOptions(OptionParser &op) {
  op.addOption("rows", OPT_INT, "0", "number of rows");
  op.addOption("cols", OPT_INT, "0", "number of cols");
  op.addOption("pyramidHeight", OPT_INT, "0", "pyramid height");
  op.addOption("resultsfile", OPT_STRING, "", "file to write results to");
}

// ****************************************************************************
// Function: RunBenchmark
//
// Purpose:
//   Executes the pathfinder benchmark
//
// Arguments:
//   resultDB: results from the benchmark are stored in this db
//   op: the options parser / parameter database
//
// Returns:  nothing, results are stored in resultDB
//
// Programmer: Kyle Spafford
// Creation: August 13, 2009
//
// Modifications:
//
// ****************************************************************************
void RunBenchmark(ResultDatabase &resultDB, OptionParser &op) {
  int device;
  hipGetDevice(&device);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device);

  int rowLen = op.getOptionInt("rows");
  int colLen = op.getOptionInt("cols");
  int pyramidHeight = op.getOptionInt("pyramidHeight");
  
  if(rowLen == 0 || colLen == 0 || pyramidHeight == 0) {
      printf("Parameters not fully specified, using preset problem size\n");
      int rowSizes[4] = {8, 16, 32, 64};
      int colSizes[4] = {4, 8, 16, 24};
      int pyramidSizes[4] = {2, 4, 8, 16};
      rows = rowSizes[op.getOptionInt("size") - 1];
      cols = colSizes[op.getOptionInt("size") - 1] * 1024 * 1024;
      pyramid_height = pyramidSizes[op.getOptionInt("size") - 1];
  } else {
      rows = rowLen;
      cols = colLen;
      pyramid_height = pyramidHeight;
  }

  printf("Row length: %d\n", rows);
  printf("Column length: %d\n", cols);
  printf("Pyramid height: %d\n", pyramid_height);

  /* --------------- pyramid parameters --------------- */
  int borderCols = (pyramid_height)*HALO;
  int smallBlockCol = BLOCK_SIZE - (pyramid_height)*HALO * 2;
  int blockCols = cols / smallBlockCol + ((cols % smallBlockCol == 0) ? 0 : 1);

  printf(
          "gridSize: [%d],border:[%d],blockSize:"
          "[%d],blockGrid:[%d],targetBlock:[%d]\n",
          cols, borderCols, BLOCK_SIZE, blockCols, smallBlockCol);

  int passes = op.getOptionInt("passes");
  for(int i = 0; i < passes; i++) {
    printf("Pass %d: ", i);
    run(borderCols, smallBlockCol, blockCols, resultDB, op);
    printf("Done.\n");
  }
}

void init(OptionParser &op) {
  data = new int[rows * cols];
  wall = new int *[rows];
  for (int n = 0; n < rows; n++) wall[n] = data + (int)cols * n;
  result = new int[cols];

  srand(SEED);

  for (int i = 0; i < rows; i++) {
    for (int j = 0; j < cols; j++) {
      wall[i][j] = rand() % 10;
    }
  }
  string resultsfile = op.getOptionString("resultsfile");
  if(resultsfile != "") {
    std::fstream fs;
    fs.open(resultsfile.c_str(), std::fstream::in);
    fs.close();
  }
}

void fatal(char *s) { fprintf(stderr, "error: %s\n", s); }

#define IN_RANGE(x, min, max) ((x) >= (min) && (x) <= (max))
#define CLAMP_RANGE(x, min, max) x = (x < (min)) ? min : ((x > (max)) ? max : x)
#define MIN(a, b) ((a) <= (b) ? (a) : (b))

__global__ void dynproc_kernel(int iteration, int *gpuWall, int *gpuSrc,
                               int *gpuResults, int cols, int rows,
                               int startStep, int border) {
  __shared__ int prev[BLOCK_SIZE];
  __shared__ int result[BLOCK_SIZE];

  int bx = blockIdx.x;
  int tx = threadIdx.x;

  // each block finally computes result for a small block
  // after N iterations.
  // it is the non-overlapping small blocks that cover
  // all the input data

  // calculate the small block size
  int small_block_cols = BLOCK_SIZE - iteration * HALO * 2;

  // calculate the boundary for the block according to
  // the boundary of its small block
  int blkX = (int)small_block_cols * (int)bx - (int)border;
  int blkXmax = blkX + (int)BLOCK_SIZE - 1;

  // calculate the global thread coordination
  int xidx = blkX + (int)tx;

  // effective range within this block that falls within
  // the valid range of the input data
  // used to rule out computation outside the boundary.
  int validXmin = (blkX < 0) ? -blkX : 0;
  int validXmax = (blkXmax > (int)cols - 1) ? (int)BLOCK_SIZE - 1 - (blkXmax - (int)cols + 1)
                                       : (int)BLOCK_SIZE - 1;

  int W = tx - 1;
  int E = tx + 1;

  W = (W < validXmin) ? validXmin : W;
  E = (E > validXmax) ? validXmax : E;

  bool isValid = IN_RANGE(tx, validXmin, validXmax);

  if (IN_RANGE(xidx, 0, (int)cols - 1)) {
    prev[tx] = gpuSrc[xidx];
  }
  __syncthreads();  // [Ronny] Added sync to avoid race on prev Aug. 14 2012
  bool computed;
  for (int i = 0; i < iteration; i++) {
    computed = false;
    if (IN_RANGE(tx, i + 1, BLOCK_SIZE - i - 2) && isValid) {
      computed = true;
      int left = prev[W];
      int up = prev[tx];
      int right = prev[E];
      int shortest = MIN(left, up);
      shortest = MIN(shortest, right);
      int index = cols * (startStep + i) + xidx;
      result[tx] = shortest + gpuWall[index];
    }
    __syncthreads();
    if (i == iteration - 1) break;
    if (computed)  // Assign the computation range
      prev[tx] = result[tx];
    __syncthreads();  // [Ronny] Added sync to avoid race on prev Aug. 14 2012
  }

  // update the global memory
  // after the last iteration, only threads coordinated within the
  // small block perform the calculation and switch on ``computed''
  if (computed) {
    gpuResults[xidx] = result[tx];
  }
}

/*
   compute N time steps
*/
int calc_path(int *gpuWall, int *gpuResult[2], int rows, int cols,
              int pyramid_height, int blockCols, int borderCols, double& kernelTime) {
  dim3 dimBlock(BLOCK_SIZE);
  dim3 dimGrid(blockCols);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float elapsedTime;

  int numStreams = 1;
  hipStream_t streams[numStreams];
  for(int s = 0; s < numStreams; s++) {
      hipStreamCreate(&streams[s]);
  }
  int src = 1, dst = 0;
  for (int t = 0; t < rows - 1; t += pyramid_height) {
    for(int s = 0; s < numStreams; s++) {
    int temp = src;
    src = dst;
    dst = temp;

#ifdef HYPERQ
    if(t == 0 && s == 0) {
        hipEventRecord(start, streams[s]);
    }
    dynproc_kernel<<<dimGrid, dimBlock, 0, streams[s]>>>(
        MIN(pyramid_height, rows - t - 1), gpuWall, gpuResult[src],
        gpuResult[dst], cols, rows, t, borderCols);
    if(t + pyramid_height >= rows - 1 && s == numStreams - 1) {
        hipDeviceSynchronize();
        hipEventRecord(stop, streams[s]);
        hipEventSynchronize(stop);
        CHECK_CUDA_ERROR();
        hipEventElapsedTime(&elapsedTime, start, stop);
        kernelTime += elapsedTime * 1.e-3;
    }
#else
    hipEventRecord(start, 0);
    dynproc_kernel<<<dimGrid, dimBlock>>>(
        MIN(pyramid_height, rows - t - 1), gpuWall, gpuResult[src],
        gpuResult[dst], cols, rows, t, borderCols);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    CHECK_CUDA_ERROR();
    hipEventElapsedTime(&elapsedTime, start, stop);
    kernelTime += elapsedTime * 1.e-3;
#endif
      }
    }
  return dst;
}

void run(int borderCols, int smallBlockCol, int blockCols, ResultDatabase &resultDB, OptionParser &op) {
  // initialize data
  init(op);

  int *gpuWall, *gpuResult[2];
  int size = rows * cols;

  CUDA_SAFE_CALL(hipMalloc((void **)&gpuResult[0], sizeof(int) * cols));
  CUDA_SAFE_CALL(hipMalloc((void **)&gpuResult[1], sizeof(int) * cols));
  CUDA_SAFE_CALL(hipMalloc((void **)&gpuWall, sizeof(int) * (size - cols)));

  // Cuda events and times
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float elapsedTime;
  double transferTime = 0.;
  double kernelTime = 0;

  hipEventRecord(start, 0);
  CUDA_SAFE_CALL(hipMemcpy(gpuResult[0], data, sizeof(int) * cols, hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy(gpuWall, data+cols, sizeof(int) * (size-cols),
             hipMemcpyHostToDevice));
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  transferTime += elapsedTime * 1.e-3; // convert to seconds

  int final_ret = calc_path(gpuWall, gpuResult, rows, cols, pyramid_height,
                            blockCols, borderCols, kernelTime);

  hipEventRecord(start, 0);
  CUDA_SAFE_CALL(hipMemcpy(result, gpuResult[final_ret], sizeof(int) * cols,
             hipMemcpyDeviceToHost));
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  transferTime += elapsedTime * 1.e-3; // convert to seconds


  string resultsfile = op.getOptionString("resultsfile");
  if(!resultsfile.empty()) {
    std::fstream fs;
    fs.open(resultsfile.c_str(), std::fstream::app);
    fs << "***DATA***" << std::endl;
    for (int i = 0; i < cols; i++) {
      fs << data[i] << " ";
    }
    fs << std::endl;
    fs << "***RESULT***" << std::endl;
    for (int i = 0; i < cols; i++) {
      fs << result[i] << " ";
    }
    fs << std::endl;
  }

  hipFree(gpuWall);
  hipFree(gpuResult[0]);
  hipFree(gpuResult[1]);

  delete[] data;
  delete[] wall;
  delete[] result;

  string atts = toString(rows) + "x" + toString(cols);
  resultDB.AddResult("Pathfinder-TransferTime", atts, "sec", transferTime);
  resultDB.AddResult("Pathfinder-KernelTime", atts, "sec", kernelTime);
  resultDB.AddResult("Pathfinder-TotalTime", atts, "sec", transferTime + kernelTime);
  resultDB.AddResult("Pathfinder-Rate_Parity", atts, "N", transferTime/kernelTime);
}
